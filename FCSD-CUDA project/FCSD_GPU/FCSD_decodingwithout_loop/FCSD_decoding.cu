#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
/*
 * this function implement fixed complexity sphere decoding
 * INPUT:
 * y: received signal
 * H: permuted propagation matrix
 * M: modulation scheme, (2: BPSK 4: QPSK, 16: 16QAM, 64: 64QAM)
 * psymbolconstellation: the symbol constellation
 * OUTPUT:
 * s: detection result
 * Eu: Euclidean distance
 */
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_complex.h>
#include <string.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<time.h>
#include"cu_complex_operation.cuh"
#include"common.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<cudaProfiler.h>
#include<hip/hip_runtime_api.h>
#define BLOCK_NUM 64
#define stride 256
__global__ void FEpath(
		hipComplex *R,  //upper triangular matrix after cholesky factorization
		hipComplex *s_hat,  //unconstrained estimation of transmitted symbol vector s
		hipComplex *s_potential_matrix,   //the matrix use to store all the solution candidates from all the blocks
		int *s_sub_index,   //full factorial index matrix
//		hipComplex *s,  //decoding results
		float *Eu,  //Euclidean distance
		int Nt,    //the number of transmit antennas
		int Nr,    //the number of receive antennas
		int M,    //modulation scheme
		int threadNum,    //number of threads
		int *list,     //the permutation list
		hipComplex *psymbolconstellation




		//the variable for test
		)//    hipComplex *Eu_norm_mini_share=array+Nr*Nt+Nt*threadNum+rho*threadNum+Nt+threadNum+Nt*threadNum+threadNum;
{
	//need to consider the resource allocation
	int tx=blockIdx.x*blockDim.x+threadIdx.x;     //if the path number is small we can allocate the kernel into one block so that we can use the shared memory
int tid=threadIdx.x;
int bid=blockIdx.x;
//allocate shared memory
	extern __shared__ hipComplex array[];
	 extern __shared__ float Eu_vector[];
    __shared__ int mini_Eu_index_temp;
    __shared__ int mini_Eu_index;
//    extern __shared__ int s_sub_index[];
	error_t error;
	int count1, count2,count3,count4,lag;
	__shared__ float d;    //the minimum distance unit between the signal constellation, the distance is usually 2d
	int rho=ceil(sqrt(float(Nt))-1);
	int pathNum=pow(float(M),float(rho));


//	int blockNum=pathNum/(threadNum);
//	int threadNum=1024;

//	float *distance;
//	int *resu
//#if __CUDA_ARCH__ >+300

//#endif
	   __shared__ hipComplex alpha, beta;
	   alpha.x=1;
	   alpha.y=0;
	   beta.x=0;
	   beta.y=0;
//	hipComplex *R_Eu_share;
//	   if(tx==1)
//	   {
/*R_share=Nr*Nt;
 * s_matrix_share: Nt*threadNumerror=hipMemcpy(d_R, R, Nr*Nt*sizeof(hipComplex),hipMemcpyHostToDevice);
//			if(error!=hipSuccess)
//			{
//				printf("hipMemcpy d_R returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
//			}
 * s_sub_share: rho*threadNum
 * s_hat_share : Nt*1
 * s_temp: threadNum*1
 * R_Eu_share:Nt*threadNum
 * Eu_norm_share : threadNum*1
// * Eu_norm_mini_share :1
 * s_mini_share:  Nt*1
 */

	hipComplex *R_share=array;   //the upper trian
	hipComplex *s_matrix_share=array+Nt*Nt; //the full expansion detection different for different path
	hipComplex *s_sub_share=array+Nt*Nt+Nt*threadNum;
    hipComplex *s_hat_share=array+Nt*Nt+Nt*threadNum+rho*threadNum;
    hipComplex *s_temp=array+Nt*Nt+Nt*threadNum+rho*threadNum+Nt;
    hipComplex *R_Eu_share=array+Nt*Nt+Nt*threadNum+rho*threadNum+Nt+threadNum;
    hipComplex *Eu_norm_share=array+Nt*Nt+Nt*threadNum+rho*threadNum+Nt+threadNum+Nt*threadNum;
//    hipComplex *s_mini_share=array+Nr*Nt+Nt*threadNum+rho*threadNum+Nt+threadNum+Nt*threadNum+threadNum;

	//in each thread there are one column sthore s_hat, s_share s_hat_share s_share_Q and s_sub_share
//	hipComplex *s_share=array+Nr*Nt+threadNum*rho+Nt;  //the single expansion before quantization
//	hipComplex *R_temp=array+Nr*Nt+threadNum*rho*+Nt+threadNum*Nt;
//	hipComplex *s_share_Q=array+Nr*Nt+threadNum*rho+(Nt+threadNum)+threadNum*Nt;//the single expansion after quantization
//    hipComplex *R_Eu_share1=array+Nr*Nt+thread
 //used to store the vector of R(s_Q-s_hat);
//    hipComplex *R_Eu_share2=array+Nr*Nt+threadNum*rho+(Nt+threadNum)+threadNum*Nt+2*threadNum*Nt;//stored in column major
//    float *distance=array+Nr*Nt*sizeof(hipComplex)+pathNum*rho*sizeof(hipComplex)+(Nt+1)*sizeof(hipComplex)+2*pathNum*Nt*sizeof(hipComplex)+Nr*pathNum*sizeof(hipComplex);
//	   }
//	   __syncthreads();
    //single expansion
    //
//    for(int i=0;i<pathNum;i+=threadNum)
//    {
    if(tid>=0&&tid<MATRIX_SIZE)
    {
//	for(count1=0;count1<Nr;count1++)
//	{
		for(count2=0;count2<MATRIX_SIZE;count2++)
		{
			R_share[IDC2D(tid,count2,Nt)]=R[IDC2D(tid,count2,Nt)];
		}
//	}

		s_hat_share[tid]=s_hat[tid];
    }
	__syncthreads();
//	if(tx==0)
//	{
//		printf("the upper triangular matrix before decoding is:\n");
//		  for(count1=0;count1<MATRIX_SIZE;count1++)
//		  {
//			  for(count2=0;count2<MATRIX_SIZE;count2++)
//			  {
//				  printf("%0.4f%+0.4fi ", R_share[IDC2D(count1,count2,MATRIX_SIZE)].x,R_share[IDC2D(count1,count2,MATRIX_SIZE)].y);
//
//			  }
//			  printf("\n");
//		  }
//		  printf("s_hat is!!!!\n");
//		  for(count1=0;count1<MATRIX_SIZE;count1++)
//		  {
//			  printf("%0.4f%+0.4fi ",s_hat_share[count1].x,s_hat_share[count1].y );
//		  }
//		  printf("kernel is working\n");
//	}
	for(count4=0;count4<5;count4++)
	{
	for(count1=0;count1<rho;count1++)
	{
	s_sub_share[IDC2D(tid,count1,rho)]=psymbolconstellation[s_sub_index[IDC2D(((blockIdx.x*blockDim.x*stride)+blockDim.x*count4+threadIdx.x),count1,rho)]];//read by column major so that it can be continue
	}
for (count1=Nt-1; count1>=0; count1--)
{
		if (count1<Nt-rho)
		{
			s_temp[tid]=s_hat_share[count1];
			for (count2=count1+1;count2<Nt; count2++)
			{
				s_temp[tid]=complex_add(s_temp[tid],complex_mulcom(complex_div(R_share[IDC2D(count1,count2,Nt)],R_share[IDC2D(count1,count1,Nt)]),(complex_sub(s_hat_share[count2],s_matrix_share[IDC2D(tid,count2,Nt)]))));
			}
//			s_share[IDC2D(tid,count1,Nt)]=R_temp[tid];

	//mapping the estimation s_share to the correlated signal constellation(to be continued)
	if(M==2)   //BPSK
	{
//     for(count1=0;count1<Nt-rho; count1++)
//     {
    	 d=sqrt(float(float(1)/float(Nt)));
//    	 s_share_Q[IDC2D(threadIdx.x,count1,Nt)].x=float(int(R_temp[threadIdx.x].x/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
//    	 s_share_Q[IDC2D(threadIdx.x,count1,Nt)].y=0;
//     }0.6565-1.0724i 0.1581+0.1581i 0.3595-0.7887i 0.1581-0.4743i

    	 if(s_temp[tid].x>0)
    	 {
    		 s_matrix_share[IDC2D(tid,count1,Nt)].x=d;
    		 s_matrix_share[IDC2D(tid,count1,Nt)].y=0;
    	 }
    	 else
    	 {
    		 s_matrix_share[IDC2D(tid,count1,Nt)].x=(-d);
    		 s_matrix_share[IDC2D(tid,count1,Nt)].y=0;
    	 }
	}
	else if(M==4)   //QPSK
	{
//		if(tx==0)//		printf("the up triangular matrix is:\n");
		//		  for(count1=0;count1<MATRIX_SIZE;count1++)
		//		  {
		//			  for(count2=0;count2<MATRIX_SIZE;count2++)
		//			  {
		//				  printf("%0.4f%+0.4fi ", R_share[IDC2D(count1,count2,MATRIX_SIZE)].x,R_share[IDC2D(count1,count2,MATRIX_SIZE)].y);
		//
		//			  }
		//			  printf("\n");
		//		  }
		//		  printf("kernel is working\n");
//		{
//       #if __CUDA_ARCH__ >=300
		int *result=(int*)malloc(sizeof(int));

		memset(result,0,sizeof(int));
		float *distance=(float*)malloc(M*sizeof(float));
		memset(distance,0,M*sizeof(int));
	    	d=sqrt(float(float(1)/float(Nt)));
           for(count2=0;count2<M;count2++)
           {
        	   switch(count2)
        	   {
        	   case 0:
        	   distance[count2]=sqrt(pow(s_temp[tid].x-(-d),2)+pow(s_temp[tid].y-0,2)); break;
        	   case 1:
        	   distance[count2]=sqrt(pow(s_temp[tid].x-0,2)+pow(s_temp[tid].y-(-d),2)); break;
        	   case 2:
        	   distance[count2]=sqrt(pow(s_temp[tid].x-(d),2)+pow(s_temp[tid].y-0,2)); break;
        	   case 3:
        	   distance[count2]=sqrt(pow(s_temp[tid].x-0,2)+pow(s_temp[tid].y-d,2)); break;
               default:
                   #if __CUDA_ARCH__ >=300
            	   printf("result error code %d\n", error);
                   #endif
            	   break;
        	   }
           }
//#if __CUDA_ARCH__ >=300
//           ret=hipblasIsamin(handle,4,distance,1,result);
//#endif
           float  mini_distance;
  	     int mini_index;
  	     mini_distance=distance[0];
  	     mini_index=1;
  	     for(count3=0;count3<M;count3++)
  	     {
  	    	if(distance[count3]<mini_distance)
  	    	{
  	    		mini_distance=distance[count3];
  	    		mini_index=count3+1;
  	    	}
  	     }



//           {
//             #if __CUDA_ARCH__ >=300
//        	   printf("hipblasIsamin failed return error code %d, line %d\n",error,__LINE__);
////        	   exit(EXIT_bbbb
//#endif
//           }

           switch (mini_index)
           {
           case 1:
        	s_matrix_share[IDC2D(tid,count1,Nt)].x=-d;
        	s_matrix_share[IDC2D(tid,count1,Nt)].y=0;
        	break;
           case 2:
           	s_matrix_share[IDC2D(tid,count1,Nt)].x=0;
           	s_matrix_share[IDC2D(tid,count1,Nt)].y=-d;
           	break;
           case 3:
           	s_matrix_share[IDC2D(tid,count1,Nt)].x=d;
           	s_matrix_share[IDC2D(tid,count1,Nt)].y=0;
           	break;
           case 4:
       	  s_matrix_share[IDC2D(tid,count1,Nt)].x=0;
       	  s_matrix_share[IDC2D(tid,count1,Nt)].y=d;
       	  break;
           default:
               #if __CUDA_ARCH__ >=300
        	   printf("result error code %d\n", error);
               #endif
           }
           free(distance);
           free(result);
	}
//#if __CUDA_ARCH__ >=300
//          hipblasDestroy(handle);
//#endif
	else if(M==16)  //16QAM
	{
//	     for(count1=0;count1<Nt-rho; count1++)
//	     {
	    	 d=sqrt(3/(2* (float)(Nt*(M-1))));
//	  s_share_Q[IDC2D(tid,count1,Nt)].x=float(int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
//	  s_share_Q[IDC2D(threadIdx.x,count1,Nt)].y=float(int(s_share[IDC2D(threadIdx.x,count1,Nt)].y/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
	     if(s_temp[tid].x<(-2*d))
	    	{
	    	  s_matrix_share[IDC2D(tid,count1,Nt)].x=(-3*d);
	    	}
	    else if(s_temp[tid].x>(2*d))
	    	{
	    	  s_matrix_share[IDC2D(tid,count1,Nt)].x=(3*d);
	    	}
	    else if(s_temp[tid].x>=0&&s_temp[tid].x<=2*d)
	    	{
	    	  s_matrix_share[IDC2D(tid,count1,Nt)].x=d;
	    	}
	    else if(s_temp[tid].x>=(-2*d)&&s_temp[tid].x<=0)
	        {
	    	  s_matrix_share[IDC2D(tid,count1,Nt)].x=(-d);
	    	}




	    if(s_temp[tid].y<(-2*d))
	    	{
	    	   s_matrix_share[IDC2D(tid,count1,Nt)].y=(-3*d);
	    	}
	   else if(s_temp[tid].y>(2*d))
	    	 {
	    	  s_matrix_share[IDC2D(tid,count1,Nt)].y=(3*d);
	         }
	   else if(s_temp[tid].y>=0&&s_temp[tid].y<=(2*d))
	        {
	    	s_matrix_share[IDC2D(tid,count1,Nt)].y=d;
	    	}
	    else if(s_temp[tid].y>=(-2*d)&&s_temp[tid].y<=0)
	       {
	        s_matrix_share[IDC2D(tid,count1,Nt)].y=(-d);
	       }
//	     }
	}
	else if(M==64)   //64QAM
		{
//	     for(count1=0;count1<Nt-rho; count1++)
//	     {
	    	 d=sqrt(3/(2* (float)(Nt*(M-1))));
//	     s_matrix_share[IDC2D(tid,count1,Nt)].x=float(int(s_matrix_share[IDC2D(threadIdx.x,count1,Nt)].x/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
//	    s_matrix_Q[IDC2D(threadIdx.x,count1,Nt)].y=float(int(s_matrix_share[IDC2D(threadIdx.x,count1,Nt)].y/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
	     }
//		}
		}
	else
	{
//			s_share[IDC2D(tid,count1,Nt)]=s_sub_share[IDC2D(tx,(Nt-count1-1),Nt)];
		s_matrix_share[IDC2D(tid,count1,Nt)]=s_sub_share[IDC2D(tid,(Nt-count1-1),rho)];
//		printf("the s_matrix_share in each thread is:\n");
//		printf("%0.4f%+0.4fi ", s_matrix_share[IDC2D(tid,count1,Nt)].x,s_matrix_share[IDC2D(tid,count1,Nt)].y);
//		printf("\n");
	}
	}
//__syncthreads();


//calculation of the Euclidean distance of all the symbol vectors in one block
Eu_vector[tid]=0;
for(count2=0;count2<Nt;count2++)
{
R_Eu_share[IDC2D(tid,count2,Nt)]=complex_sub(s_matrix_share[IDC2D(tid,count2,Nt)],s_hat_share[count2]);
}

for(count2=0;count2<Nt;count2++)
{
Eu_norm_share[tid]=beta;
for(count3=0;count3<Nt;count3++)
{
Eu_norm_share[tid]=complex_add(Eu_norm_share[tid],complex_mulcom(R_share[IDC2D(count2,count3,Nt)],R_Eu_share[IDC2D(tid,count2,Nt)]));
}
		Eu_vector[tid]=Eu_vector[tid]+pow(Eu_norm_share[tid].x,2)+pow(Eu_norm_share[tid].y,2);
}

__syncthreads();
//find out the symbol vector candidate with minimum Euclidean distance in one block
if(tid==0)
{
//	printf("let go Eu_vector!!\n");
//	for(count1=0;count1<MATRIX_SIZE;count1++)
//	{
//		printf("%0.4f ", Eu_vector[count1]);
//	}
       __shared__ float  mini_Eu;
	     mini_Eu=Eu_vector[0];
         mini_Eu_index=0;
	     for(count3=0;count3<threadNum;count3++)
	     {
	    	if(Eu_vector[count3]<mini_Eu)
	    	{
	    		mini_Eu=Eu_vector[count3];
	    		mini_Eu_index_temp=count3;
	    	}
	     }
lag=0;
if(count4==0)
{
Eu[bid]=mini_Eu;
mini_Eu_index=mini_Eu_index_temp;
lag=1;
}
else
{
if(mini_Eu<Eu[bid])
{
	Eu[bid]=mini_Eu;
	mini_Eu_index=mini_Eu_index_temp;
	lag=1;
}
}

}
//__syncthreads();
if(tid>=0&&tid<=(MATRIX_SIZE-1))
{
if(lag==1)
{
	s_potential_matrix[IDC2D(bid,(list[tid]-1),Nt)]=s_matrix_share[IDC2D(mini_Eu_index,(MATRIX_SIZE-tid-1),Nt)];
}
//	s_potential_matrix[IDC2D(bid,(list[tid]-1),Nt)].x=1;
//	s_potential_matrix[IDC2D(bid,(list[tid]-1),Nt)].y=1;
}
__syncthreads();

	}
//}
//if(tid==0)
//{
////	printf("the s_potential_matrix is: \n");
////	for(count1=0;count1<threadNu;count1++)	int stride=512;
////	{
//
////printf("the s_share_matrix after iteration is:\n");
////for(count1=0;count1<threadNum;count1++)
////{
////	for(count2=0;count2<Nt;count2++)
////	{
////	printf("%0.4f%+0.4fi ", s_matrix_share[IDC2D(count1,count2,Nt)].x, s_matrix_share[IDC2D(count1,count2,Nt)].y);
////	}
////	printf("\n");
////}
//
//		for(count2=0;count2<Nt;count2++)
//		{
//		printf("the %d th row of the s_potential_matrix is :%0.4f%+0.4fi ", bid, s_potential_matrix[IDC2D(bid,count2,Nt)].x, s_potential_matrix[IDC2D(bid,count2,Nt)].y);
//		}
//		printf("\n");
////	}
//	printf("the miniEuindex is:\n");//error=hipMemcpy(pR, d_pR, Nt*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
//	//      	    if(error!=hipSuccess)
//	//      	     {
//	//      	    	 printf("hipMemcpy pR returned error code %d, line %d\n", error, __LINE__);
//	//      	    	  exit(EXIT_FAILURE);
//	//      	      }
//	printf("%d ", mini_Eu_index);
//	printf("the Eu_vector is:\n");
//	for(count1=0;count1<threadNum;count1++)
//	{
//		printf("%0.4f ", Eu_vector[count1]);
//	}
//	printf("\n");
//
//			printf("the upper triangular matrix after decoding is:\n");
//			  for(count1=0;count1<MATRIX_SIZE;count1++)
//			  {
//				  for(count2=0;count2<MATRIX_SIZE;count2++)
//				  {
//					  printf("%0.4f%+0.4fi ", R_share[IDC2D(count1,count2,MATRIX_SIZE)].x,R_share[IDC2D(count1,count2,MATRIX_SIZE)].y);
//
//				  }
//				  printf("\n");
//			  }
//			  printf("FEpath kernel is working\n");
//}

}
//	printf("the FEpath is working\n");
//	for(count1=0;count1<MATRIX_SIZE;count1++)
//	{
//		for(count2=0;count2<MATRIX_SIZE;count2++)
//		{
//			R[IDC2D(count1,count2,Nt)].x=1;
//			R[IDC2D(count1,count2,Nt)].y=1;
//		}
//	}//error=hipMemcpy(pR, d_pR, Nt*Nt*sizeof(hipComplex),cudaMemcpyDeviceToH//	if(tid==0)
//	{ost);
//      	    if(error!=hipSuccess)
//      	     {
//      	    	 printf("hipMemcpy pR returned error code %d, line %d\n", error, __LINE__);
//      	    	  exit(EXIT_FAILURE);
//      	      }
//}
//__syncthreads();



//if(tx==0)
//{
//printf("the detected symbol vector is:\n");
//for(count1=0;count1<MATRIX_SIZE;count1++)
//{
//	printf("%0.4f%+0.4f ", s[count1].x, s[count1].y);
//}//error=hipMemcpy(pR, d_pR, Nt*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
//      	    if(error!=hipSuccess)
//      	     {
//      	    	 printf("hipMemcpy pR returned error code %d, line %d\n", error, __LINE__);
//      	    	  exit(EXIT_FAILURE);
//      	      }
//}
//}

//host
void FCSD_decoding(
		hipComplex *d_R,  //upper triangular matrix after cholesky factorization store in device side
//		hipComplex *s_sub, //the sub brute force rho vector matrix
		hipComplex *d_s_hat,  //unconstrained estimation of transmitted symbol vector s
		hipComplex *s_kernel,  //quantization of estimation ,decoding results
//		hipComplex *Eu,  //Euclidean distance
		int Nt,    //the number of transmit antennas
		int Nr,    //the number of receive antennas
		int M,    //modulation scheme
		int *list,   //the permutation list
		hipComplex *psymbolconstellation //the symbol constellation
		)
{
//brute force search determine the vector results of the full expansion
	int rho=ceil(sqrt(Nt)-1);
//	hipComplex *ss;
//	ss=(hipComplex*)malloc(MATRIX_SIZE*sizeof(hipComplex));
//	hipComplex *s_sub;
//	s_sub=(hipComplex*)malloc(pow(M,rho)*rho*sizeof(hipComplex));   //all the possible full expansion sub vector
	int  pathNum;
	pathNum=pow(M,rho);
	int *d_list,*d_s_sub_index;
	int *s_sub_index=(int*)calloc(1,rho*pow(M,rho)*sizeof(int));
	fullfact(rho,M,s_sub_index);    //get  the indexes of all the possible rho length symbol vectors
	int blockNum=BLOCK_NUM;   //determined by the path number
//	int pathNum=pow(M,rho);  //number of search path
	int threadNum=pathNum/(blockNum*stride); //determined by the path number
	float *Eu,*d_Eu;
	Eu=(float*)calloc(1,blockNum*sizeof(float));
	hipComplex *s_potential_matrix=(hipComplex*)calloc(1,blockNum*Nt*sizeof(hipComplex));
	hipComplex  *d_s_potential_matrix,*d_psymbolconstellation;
	int count1;
//	int *j;
//	j=(int*)malloc(sizeof(int));
	hipblasHandle_t handle;
		hipblasStatus_t ret;
		hipError_t error;
		ret=hipblasCreate(&handle);
	    if (ret != HIPBLAS_STATUS_SUCCESS)
	    {
//	        printf("hipblasCreate returned error code %d, line(%d)\n", ret, __LINE__);
//	        exit(EXIT_FAILURE);
	    }
	    error=hipMalloc((void**) &d_s_sub_index, rho*pow(M,rho)*sizeof(int));
	    if(error!=hipSuccess)
	    {
//	    	printf("hipMalloc d_s_sub_index returned error code %d, line %d\n", error, __LINE__);
//	    	exit(EXIT_FAILURE);
	    }
//	error=hipMalloc((void**) &d_R, Nr*Nt*sizeof(hipComplex));
//			if(error!=hipSuccess)
//			{
//				printf("hipMalloc d_R returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
//			}
//			error=hipMalloc((void**) &d_s_sub, rho*sizeof(hipComplex));
//			if(error!=hipSuccess)
//			{
//				printf("hipMalloc d_s_sub returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
//			}
//			error=hipMalloc((void**) &d_s_hat, Nt*sizeof(hipComplex));
//			if(error!=hipSuccess)
//			{
////				printf("hipMalloc d_s_hat returned error code %d, line %d\n", error, __LINE__);
////				exit(EXIT_FAILURE);
//			}
			error=hipMalloc((void**) &d_s_potential_matrix, blockNum*Nt*sizeof(hipComplex));
			if(error!=hipSuccess)
			{
//				printf("hipMalloc d_R returned error code %d,	 line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
			}
			error=hipMalloc((void**) &d_list, MATRIX_SIZE*sizeof(int));
			if(error!=hipSuccess)
			{
//				printf("hipMalloc d_list returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
			}
			error=hipMalloc((void**) &d_Eu, blockNum*sizeof(float));
			if(error!=hipSuccess)
			{
//				printf("hipMalloc d_Eu returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
			}
         error=hipMalloc((void**) &d_psymbolconstellation, M*sizeof(hipComplex));
	       if(error!=hipSuccess)
	          {
//	        	printf("hipMalloc d_psymbolconstellation returned error code %d, line %d\n", error, __LINE__);
//		        exit(EXIT_FAILURE);
	           }

//			error=hipMemcpy(d_R, R, Nr*Nt*sizeof(hipComplex),hipMemcpyHostToDevice);
//			if(error!=hipSuccess)
//			{
//				printf("hipMemcpy d_R returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
//			}
//			error=hipMemcpy(d_s_sub, s_sub, rho*sizeof(hipComplex),hipMemcpyHostToDevice);
//			if(error!=hipSuccess)
//			{
//				printf("hipMemcpy d_s_sub returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
//			}
//			error=hipMemcpy(d_s_hat, s_hat, Nt*sizeof(hipComplex),hipMemcpyHostToDevice);
//			if(error!=hipSuccess)
//			{
//				printf("hipMemcpy d_s_hat returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
//			}
//			error=hipMemcpy(d_s_potential_matrix, s_potential_matrix, blockNum*Nt*sizeof(hipComplex),hipMemcpyHostToDevice);
//			if(error!=hipSuccess)
//			{
////				printf("hipMemcpy d_s_potential_matrix returned error code %d, line %d\n", error, __LINE__);
////				exit(EXIT_FAILURE);
//			}
//			error=hipMemcpy(d_Eu, Eu, (blockNum)*sizeof(float),hipMemcpyHostToDevice);
//			if(error!=hipSuccess)
//			{
////				printf("hipMemcpy d_Eu returned error code %d, line %d\n", error, __LINE__);
////				exi256t(EXIT_FAILURE);
//			}
			error=hipMemcpy(d_psymbolconstellation, psymbolconstellation, M*sizeof(hipComplex),hipMemcpyHostToDevice);
						if(error!=hipSuccess)
						{
//							printf("hipMemcpy d_psynbolconstellation returned error code %d, line %d\n", error, __LINE__);
//							exit(EXIT_FAILURE);
						}
		error=hipMemcpy(d_s_sub_index, s_sub_index, rho*int(pow(M,rho))*sizeof(int),hipMemcpyHostToDevice);
				if(error!=hipSuccess)
					{
//						printf("hipMemcpy d_psynbolconstellation returned error code %d, line %d\n", error, __LINE__);
//							exit(EXIT_FAILURE);
				}
				error=hipMemcpy(d_list, list, Nt*(sizeof(int)),hipMemcpyHostToDevice);
					if(error!=hipSuccess)
					{
//					printf("hipMemcpy d_s_kernel returned error code %d, line %d\n", error, __LINE__);
//					exit(EXIT_FAILURE);
				    }
			//add the psymbolconstellation
//				size_t heapsize;
//				heapsize=1024*sizeof(float);
				int sharedMem;
//				sharedMem=(Nr*Nt+Nt*threadNum+rho*threadNum+Nt+threadNum+Nt*threadNum+threadNum)*sizeof(hipComplex)+threadNum*sizeof(float);
				sharedMem=6000*sizeof(hipComplex);
//	hipDeviceSetLimit(hipLimitMallocHeapSize, heapsize);
//	int Eu_num;
//	memset(Eu_num,0,sizeof(int));
//	Eu_num=0;
   float duration;
   clock_t start, end;
//   hipProfilerStart();
   start=clock();
	FEpath<<<blockNum, threadNum,sharedMem>>>(d_R, d_s_hat,d_s_potential_matrix,d_s_sub_index, d_Eu, Nr, Nt, M,threadNum,d_list,d_psymbolconstellation);
	end=clock();
//	hipProfilerStop();
	duration=double(end-start);
	printf("hey %0.4f ", duration);
	printf("\n");
	error=hipDeviceSynchronize();
		if(error!=hipSuccess)
		{
//		printf("hipDeviceSynchronize returned error code %d, line %d\n", error, __LINE__);
//				 	exit(EXIT_FAILURE);
		}
		printf("%s\n",hipGetErrorString(hipGetLastError()));
//	printf("Eu_num is %d", Eu_num);
    error=hipMemcpy(s_potential_matrix,d_s_potential_matrix,blockNum*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
    if(error!=hipSuccess)
    {
//   	printf("s_potential_matrix returned error code %d, line %d\n", error, __LINE__);
//    			 	exit(EXIT_FAILURE);
   	}
    printf("all the potential symbol vector is:\n");
    for(count1=0;count1<blockNum;count1++)
    {
    	for(int count2=0;count2<Nt;count2++)
    	{
    		printf("%0.4f%+0.4fi ", s_potential_matrix[IDC2D(count1,count2,Nt)].x,s_potential_matrix[IDC2D(count1,count2,Nt)].y);
    	}
    	printf("\n");
    }
//    error=hipMemcpy(R,d_R,Nt*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
//    printf("the test upper triangular matrix in kernel is:\n");
//    for(count1=0;count1<Nt;count1++)
//       {
//       	for(int count2=0;count2<Nt;count2++)
//       	{
//       		printf("%0.4f%+0.4fi ", R[IDC2D(count1,count2,Nt)].x,R[IDC2D(count1,count2,Nt)].y);
//       	}
//       	printf("\n");
//       }

    error=hipMemcpy(Eu,d_Eu,blockNum*sizeof(float),hipMemcpyDeviceToHost);
    if(error!=hipSuccess)
    {
//   	printf("Eu returned error code %d, line %d\n", error, __LINE__);
//    			 	exit(EXIT_FAILURE);
   	}


 //fine out the symbol vector index among all the block output Euclidean distance
    int Eu_mini_index=0;
    float Eu_mini_value=Eu[0];
    for(count1=0;count1<blockNum;count1++)
    {
      if(Eu[count1]<Eu_mini_value)
      {
    	  Eu_mini_value=Eu[count1];
    	  Eu_mini_index=count1;
      }
    }
    for(count1=0;count1<Nt;count1++)
    {
     s_kernel[count1]=s_potential_matrix[IDC2D(Eu_mini_index,count1,Nt)];
    }

//    	for(int count2=0;count2<MATRIX_SIZE;count2++)
//    	{
//    		printf("%0.4f%+0.4f ", s_share_matrix[IDC2D(count1,count2,MATRIX_SIZE)].x, s_share_matrix[IDC2D(count1,count2,MATRIX_SIZE)].y);
//    	}
//    	printf("\n");
//    }
//       error=hipMemcpy(s_hat,d_s_hat,Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
          if(error!=hipSuccess)
          			   	{
//          			  	printf("s_hat returned error code %d, line %d\n", error, __LINE__);
          //			 	exit(EXIT_FAILURE);
          			   	}
//          for(count1=0;count1<pathNum;count1++)
//          {
//          printf("the unconstrained estimation is:\n");
//          	for(int count2=0;count2<MATRIX_SIZE;count2++)
//          	{
//          		printf("%0.4f%+0.4fi ", s_hat[count2].x, s_hat[count2].y);
//          	}
//          	printf("\n");
//          }

				ret=hipblasDestroy(handle);
			    if (ret != HIPBLAS_STATUS_SUCCESS)
			    {
//			        printf("hipblasDestroy returned error code %d, line(%d)\n", ret, __LINE__);
//			        exit(EXIT_FAILURE);
			    }
			    printf("the s_kernel is :\n");
			    for(count1=0;count1<Nt;count1++)
			    {
			    	printf("%0.4f%+0.4fi ", s_kernel[count1].x, s_kernel[count1].y);
			    }
			    printf("\n");

			   	free(s_sub_index);
			   	hipFree(d_s_sub_index);
			   	hipFree(d_list);
			   	free(Eu);
			   	hipFree(d_Eu);
			   	free(s_potential_matrix);
			   	hipFree(d_s_potential_matrix);
			   	hipFree(d_psymbolconstellation);
//			   	hipFree(d_R);

}

