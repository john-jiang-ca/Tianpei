#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
//#include <assert.h>
//includes system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<math.h>
//#include<hip/hip_runtime.h>
// includes CUDA
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
//#include<cutil.h>
//includes project
//#include<hip/hip_runtime_api.h>
//#include<helper_functions.h>
#include"cu_complex_operation.cuh"
#include"common.h"
#include<hip/hip_complex.h>
#include<time.h>
#include<hip/hip_runtime_api.h>
#define BLOCKNUM 32
 __global__ void chol_kernel_optimized
 (
		 hipComplex *R
 )
 {
//	 int MATRIX_SIZE=N1;
//	 extern __shared__ hipComplex R[];
      hipComplex pivot;
      hipComplex update;
	 int count1,count2,count3;
	 int tid=threadIdx.x;
//	 bid=blockIdx.x;
//	 tid=threadIdx.x;
	  hipComplex zero;
	 zero.x=0;
	 zero.y=0;
	 //cholesky factorization row by row
	 for (count1=0;count1<MATRIX_SIZE;count1++)
	 {
//		 R[tid]=matrix[IDC2D(count1,tid,MATRIX_SIZE)];
		 //pivoting step
			if(tid==0)
			{
				if(count1==0)
				{
					R[IDC2D(count1,count1,MATRIX_SIZE)].x=sqrt(R[IDC2D(count1,count1,MATRIX_SIZE)].x);
					R[IDC2D(count1,count1,MATRIX_SIZE)].y=0;
					printf("the first element is %0.4f%+0.4fi : \n",R[IDC2D(count1,count1,MATRIX_SIZE)].x,R[IDC2D(count1,count1,MATRIX_SIZE)].y );
				}
				else
				{
				pivot=zero;
				for(count2=0;count2<count1;count2++)
				{
			 pivot=complex_add(pivot,complex_mulcom(complex_conjugate(R[IDC2D(count2,count1,MATRIX_SIZE)]),R[IDC2D(count2,count1,MATRIX_SIZE)]));
				}
				 R[IDC2D(count1,count1,MATRIX_SIZE)]=complex_sub(R[IDC2D(count1,count1,MATRIX_SIZE)],pivot);
				 R[IDC2D(count1,count1,MATRIX_SIZE)].x=sqrt(R[IDC2D(count1,count1,MATRIX_SIZE)].x);
				 R[IDC2D(count1,count1,MATRIX_SIZE)].y=0;
//				 printf("the  %d diagonal elements is %0.4f%+0.4fi:\n",count1,R[IDC2D(count1,count1,MATRIX_SIZE)].x,R[IDC2D(count1,count1,MATRIX_SIZE)].y );
			}
			}
           __syncthreads();
// update the off-diagonal elements
				if(tid>count1)
				{
				  update=zero;
					for(count2=0;count2<count1;count2++)
					{
					update=complex_add(update,complex_mulcom(complex_conjugate(R[IDC2D(count2,count1,MATRIX_SIZE)]),R[IDC2D(count2,tid,MATRIX_SIZE)]));
					}
		R[IDC2D(count1,tid,MATRIX_SIZE)]=complex_sub(R[IDC2D(count1,tid,MATRIX_SIZE)],update);
		R[IDC2D(count1,tid,MATRIX_SIZE)]=complex_div(R[IDC2D(count1,tid,MATRIX_SIZE)],complex_conjugate(R[IDC2D(count1,count1,MATRIX_SIZE)]));
				}
				else if(tid<count1)
				{
					R[IDC2D(count1,tid,MATRIX_SIZE)]=zero;
				}
		      __syncthreads();
				printf("the %d %d element is %0.4f%+0.4fi ", count1,tid, R[IDC2D(count1,tid,MATRIX_SIZE)].x,R[IDC2D(count1,tid,MATRIX_SIZE)].y);
				printf("\n");
			}

//	 if(tid>=0&&tid<MATRIX_SIZE)
//	 {
//		 for(count1=0;count1<MATRIX_SIZE;count1++)
//		 {
//		 matrix[IDC2D(count1,tid,MATRIX_SIZE)]=R[IDC2D(count1,tid,MATRIX_SIZE)];
//		 }
//	 }
//	 __syncthreads();

 }


void chol_without_onchip(hipComplex *d_U
)
{


	//int MATRIX_SIZE=sizeof(U)/;
	//int m=U->size2;
//	int count;
	int count1;
	int count2;
//int MATRIX_SIZE=N1;
//allocate computation space
	hipError_t error;
//	hipComplex *d_U;
//	error=hipMalloc((void**) &d_U, MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
//	if(error !=hipSuccess){
//		printf("hipMalloc d_U returned error code %d, line(%d)\n", error, __LINE__);
//	}
	//data transmission from CPU to GPU


	//data transmission from CPU to GPU


//	int sharedMem=MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex);
	//dim3 BlockId;
	clock_t start, end;
	start=clock();
		int threadNum=MATRIX_SIZE;
		int blockNum=1;
		int sharedMem=2000*sizeof(hipComplex);
//		int *threadID;
//		int *d_threadID;
		hipProfilerStart();
//		threadID=(int*)malloc(sizeof(int)*threadNum);
//		hipMalloc((void**)&d_threadID, threadNum*sizeof(int));
//		hipMemcpy(d_threadID,threadID,sizeof(int)*threadNum,hipMemcpyHostToDevice);
//		error=hipMemcpy(d_U, U, sizeof(hipComplex)*MATRIX_SIZE*MATRIX_SIZE, hipMemcpyHostToDevice );
//		if(error!=hipSuccess){
//			printf("hipMemcpy U to d_U returned error code %d, line(%d)\n", error, __LINE__);
//		}

		chol_kernel_optimized<<<blockNum,threadNum,sharedMem>>>(d_U);
//		hipError_t error;
		error=hipDeviceSynchronize();
		if(error!=hipSuccess)
		{
		printf("%s\n", hipGetErrorString(hipGetLastError()));
		}
//		hipMemcpy(threadID, d_threadID, sizeof(int)*threadNum, hipMemcpyDeviceToHost );
//		for(count1=0;count1<threadNum; count1++)
//		{
//		printf("%d", threadID[count1]);
//		printf("\n");
//		}
//		printf("%d",sizeof(hipComplex));
//		error=hipMemcpy(pR,d_U,sizeof(hipComplex)*MATRIX_SIZE*MATRIX_SIZE, hipMemcpyDeviceToHost );
//		if(error!=hipSuccess)
//		{
//			printf("hipMemcpy d_U to U returned error code %d, line(%d)\n", error, __LINE__);
//		}
		hipProfilerStop();
//	}
	end=clock();
//	 *durationD=(double)(end-start)/CLOCKS_PER_SEC;
	//data transmission from GPU to CPU
//	for(count1=0; count1<MATRIX_SIZE; count1++)
//	{
//		for(count2=0; count2<MATRIX_SIZE; count2++)
//		{
//			pR[count1*MATRIX_SIZE+count2]=U[count1*MATRIX_SIZE+count2];
//
//
//		}
//	}
//	U[0].x=*m;
//	pR[0].y=*m;
//    free(y);
//    free(m);
//    free(threadID);
//    hipFree(d_threadID);

}

