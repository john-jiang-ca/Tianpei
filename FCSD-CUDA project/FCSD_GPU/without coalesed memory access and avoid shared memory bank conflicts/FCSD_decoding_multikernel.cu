#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
/*
 * this function implement fixed complexity sphere decoding
 * INPUT:
 * y: received signal
 * H: permuted propagation matrix
 * M: modulation scheme, (2: BPSK 4: QPSK, 16: 16QAM, 64: 64QAM)
 * psymbolconstellation: the symbol constellation
 * OUTPUT:
 * s: detection result
 * Eu: Euclidean distance
 */
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_complex.h>
#include <string.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<time.h>
#include"cu_complex_operation.cuh"
#include"common.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<cudaProfiler.h>
#include<hip/hip_runtime_api.h>
#include<hip/device_functions.h>
#define threadNum 1024
#define blockNum 1
#define stride 1
/*
 * in this version I applied colesced memory accesss to all the vector and matrix, with all the matrix stored in row major different threads reading one column
 * with all the matrix stored in column major, different threads reading one row
 * uitilize consecutive computation power
 */
__constant__ hipComplex d_R[MATRIX_SIZE*MATRIX_SIZE],d_psymbolconstellation[16],d_constant_shat[MATRIX_SIZE];
__constant__ int d_list[MATRIX_SIZE];
__global__ void FEpath(
//		hipComplex *d_R,  //upper triangular matrix after cholesky factorization
//		hipComplex *d_constant_shat,  //unconstrained estimation of transmitted symbol vector s
//		hipComplex *s_matrix_share,
		hipComplex *s_potential_matrix,   //the matrix use to store all the solution candidates from all the blocks
		int *s_sub_index,   //full factorial index matrix
		int rho,
		int pathNum,
//		hipComplex *s,  //decoding results
		float *Eu,  //Euclidean distance
//		int pitch_R,    //the number of transmit antennas
//		int pitch_index,    //the number of receive antennas
//		int pitch_p,
		int M,    //modulation scheme
//		int threadNum,    //number of threads
//		int *d_list,     //the permutation list
//		hipComplex *d_psymbolconstellation,
		int index





		)
{
	//need to consider the resource allocation
	int tx=blockIdx.x*blockDim.x+threadIdx.x;     //if the path number is small we can allocate the kernel into one block so that we can use the shared memory
int tid=threadIdx.x;
int Nt=MATRIX_SIZE;

//allocate shared memory
	extern __shared__ hipComplex array[];

	error_t error;
	int count1, count2,count3,count4;
	__shared__ float d;    //the minimum distance unit between the signal constellation, the distance is usually 2d
	__shared__ hipComplex alpha, beta;
	alpha.x=1;alpha.y=0; beta.x=0; beta.y=0;
	__shared__ hipComplex s_hat_share[MATRIX_SIZE];
	__shared__ hipComplex R_share[MATRIX_SIZE*MATRIX_SIZE];
//    __shared__ hipComplex s_temp[threadNum];
	hipComplex s_temp;
	hipComplex Eu_norm_share;
//    __shared__ hipComplex Eu_norm_share[threadNum];
    hipComplex *R_Eu_share=(hipComplex*)malloc(Nt*sizeof(hipComplex));
    if(tid>=0&&tid<MATRIX_SIZE)
    {

		for(count1=0;count1<MATRIX_SIZE;count1++)
		{
			R_share[IDC2D(tid,count1,MATRIX_SIZE)]=d_R[IDC2D(tid,count1,MATRIX_SIZE)];
		}


		s_hat_share[tid]=d_constant_shat[tid];
    }

	__syncthreads();

	Eu[blockNum*threadNum*index+tx]=0;
for (count1=Nt-1; count1>=0; count1--)
{

		if (count1<Nt-rho)
		{
			s_temp=s_hat_share[count1];
            #pragma unroll
			for (count2=count1+1;count2<Nt; count2++)
			{
				s_temp=complex_add(s_temp,complex_mulcom(complex_div(R_share[IDC2D(count1,count2,MATRIX_SIZE)],R_share[IDC2D(count1,count1,MATRIX_SIZE)]),(complex_sub(s_hat_share[count2],s_potential_matrix[IDC2D(count2,(index*blockNum*threadNum+tx),pathNum/4)]))));
			}
	if(M==2)   //BPSK
	{

    	 d=sqrt(float(float(1)/float(Nt)));
    	 if(s_temp.x>0)
    	 {
    		  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=d;
    		  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=0;
    	 }
    	 else
    	 {
    		  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=(-d);
    		  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=0;
    	 }
	}
	else if(M==4)   //QPSK
	{
		int *result=(int*)malloc(sizeof(int));

		memset(result,0,sizeof(int));
		float *distance=(float*)malloc(M*sizeof(float));
		memset(distance,0,M*sizeof(int));
	    	d=sqrt(float(float(1)/float(Nt)));
           for(count2=0;count2<M;count2++)
           {
        	   switch(count2)
        	   {
        	   case 0:
        	   distance[count2]=sqrt(powf(s_temp.x-(-d),2)+pow(s_temp.y-0,2)); break;
        	   case 1:
        	   distance[count2]=sqrt(powf(s_temp.x-0,2)+pow(s_temp.y-(-d),2)); break;
        	   case 2:
        	   distance[count2]=sqrt(pow(s_temp.x-(d),2)+pow(s_temp.y-0,2)); break;
        	   case 3:
        	   distance[count2]=sqrt(pow(s_temp.x-0,2)+pow(s_temp.y-d,2)); break;
               default:
                   #if __CUDA_ARCH__ >=300
            	   printf("result error code %d\n", error);
                   #endif
            	   break;
        	   }
           }
           float  mini_distance;
  	     int mini_index;
  	     mini_distance=distance[0];
  	     mini_index=1;
  	     for(count3=0;count3<M;count3++)
  	     {
  	    	if(distance[count3]<mini_distance)
  	    	{
  	    		mini_distance=distance[count3];
  	    		mini_index=count3+1;
  	    	}
  	     }

           switch (mini_index)
           {
           case 1:
        	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=-d;
        	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=0;
        	break;
           case 2:
           	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=0;
           	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=-d;
           	break;
           case 3:
           	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=d;
           	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=0;
           	break;
           case 4:
       	   s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=0;
       	   s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=d;
       	  break;
           default:
               #if __CUDA_ARCH__ >=300
        	   printf("result error code %d\n", error);
               #endif
           }
           free(distance);
           free(result);
	}
	else if(M==16)  //16QAM
	{

	    	 d=sqrt(float(3)/(2* (float)(Nt*(M-1))));
	     if(s_temp.x<(-2*d))
	    	{
	    	   s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=(-3*d);
	    	}
	    else if(s_temp.x>(2*d))
	    	{
	    	   s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=(3*d);
	    	}
	    else if(s_temp.x>=0&&s_temp.x<=2*d)
	    	{
	    	   s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=d;
	    	}
	    else if(s_temp.x>=(-2*d)&&s_temp.x<=0)
	        {
	    	   s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].x=(-d);
	    	}




	    if(s_temp.y<(-2*d))
	    	{
	    	    s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=(-3*d);
	    	}
	   else if(s_temp.y>(2*d))
	    	 {
	    	   s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=(3*d);
	         }
	   else if(s_temp.y>=0&&s_temp.y<=(2*d))
	        {
	    	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=d;
	    	}
	    else if(s_temp.y>=(-2*d)&&s_temp.y<=0)
	       {
	         s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)].y=(-d);
	       }
	}
	else if(M==64)   //64QAM
		{

	    	 d=sqrt(3/(2* (float)(Nt*(M-1))));
	     }
		}
	else
	{
		 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)]=d_psymbolconstellation[s_sub_index[IDC2D((Nt-count1-1),(index*blockNum*threadNum+tx),pathNum/4)]];

	}
		R_Eu_share[count1]=complex_sub( s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pathNum/4)],s_hat_share[count1]);
		Eu_norm_share=beta;
        #pragma unroll
		for(count3=count1;count3<MATRIX_SIZE;count3++)
		{
		Eu_norm_share=complex_add(Eu_norm_share,complex_mulcom(R_share[IDC2D(count1,count3,MATRIX_SIZE)],R_Eu_share[count3]));
		}
		Eu[(index*blockNum*threadNum+tx)]=Eu[(index*blockNum*threadNum+tx)]+pow(Eu_norm_share.x,2)+pow(Eu_norm_share.y,2);

	}
__syncthreads();
free(R_Eu_share);


}

//host

void FCSD_decoding(
		hipComplex *R,  //upper triangular matrix after cholesky factorization store in device side
//		hipComplex *s_sub, //the sub brute force rho vector matrix
		hipComplex *d_s_hat,  //unconstrained estimation of transmitted symbol vector s
		hipComplex *s_kernel,  //quantization of estimation ,decoding results
//		hipComplex *Eu,  //Euclidean distance
		int Nt,    //the number of transmit antennas
		int Nr,    //the number of receive antennas
		int M,    //modulation scheme
		int *list,   //the permutation list
		hipComplex *psymbolconstellation //the symbol constellation
		)
{
//brute force search determine the vector results of the full expansion
	int rho=ceil(sqrt(Nt)-1);
	int count1,count2;
//	hipComplex *ss;
//	ss=(hipComplex*)malloc(MATRIX_SIZE*sizeof(hipComplex));
//	hipComplex *s_sub;
//	s_sub=(hipComplex*)malloc(pow(M,rho)*rho*sizeof(hipComplex));   //all the possible full expansion sub vector
	int  pathNum;
	pathNum=pow(M,rho);
//	int *d_s_sub_index;
	int *s_sub_index=(int*)calloc(1,rho*pow(M,rho)*sizeof(int));
	fullfact(rho,M,s_sub_index);    //get  the indexes of all the possible rho length symbol vectors
//	int blockNum=BLOCK_NUM;   //determined by the path number
//	int pathNum=pow(M,rho);  //number of search path
//	int threadNum=ceil(pathNum/(blockNum*stride)); //determined by the path number
    float *Eu;
	Eu=(float*)calloc(1,blockNum*sizeof(float));
//	hipComplex *s_potential_matrix=(hipComplex*)calloc(1,pathNum*Nt*sizeof(hipComplex));
	hipComplex *s_hat=(hipComplex*)calloc(1,Nt*sizeof(hipComplex));
//	hipComplex  *d_s_potential_matrix;
	hipMemcpy(s_hat,d_s_hat,Nt*sizeof(MATRIX_SIZE),hipMemcpyDeviceToHost);

//	int *j;
//	j=(int*)malloc(sizeof(int));
	hipblasHandle_t handle;
		hipblasStatus_t ret;
		hipError_t error;
		size_t pitch_R,pitch_potential,pitch_index;
		ret=hipblasCreate(&handle);
//	    error=hipMalloc((void**) &d_R, MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
//	    error=hipMalloc((void**) &d_s_sub_index, rho*pathNum*sizeof(int));
//		error=hipMalloc((void**) &d_s_potential_matrix, pathNum*Nt*sizeof(hipComplex));
//		error=hipMalloc((void**) &d_list, MATRIX_SIZE*sizeof(int));
//		error=hipMalloc((void**) &d_Eu, pathNum*sizeof(float));
//        error=hipMalloc((void**) &d_psymbolconstellation, M*sizeof(hipComplex));
        clock_t start, end;
        start=clock();
        hipComplex *R_constant=(hipComplex*)calloc(1,MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
    	hipMemcpyToSymbol(HIP_SYMBOL(d_R), R, MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex),0,hipMemcpyHostToDevice);
    	hipMemcpyFromSymbol(R_constant,HIP_SYMBOL(d_R),MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex),0,hipMemcpyDeviceToHost);
    	printf("cao ni ma!!\n");
    	for(count1=0;count1<MATRIX_SIZE;count1++)
    	{
    		for(int count2=0;count2<MATRIX_SIZE;count2++)
    		{
    			printf("%0.4f%+0.4fi ", R_constant[IDC2D(count1,count2,MATRIX_SIZE)].x,R_constant[IDC2D(count1,count2,MATRIX_SIZE)].y);
    		}
    		printf("\n");
    	}
    	hipMemcpyToSymbol(HIP_SYMBOL(d_psymbolconstellation), psymbolconstellation, M*sizeof(hipComplex),0,hipMemcpyHostToDevice);
    	hipMemcpyToSymbol(HIP_SYMBOL(d_constant_shat), s_hat, MATRIX_SIZE*sizeof(hipComplex),0,hipMemcpyHostToDevice);
    	hipMemcpyToSymbol(HIP_SYMBOL(d_list), list, MATRIX_SIZE*sizeof(int),0,hipMemcpyHostToDevice);
//        error=hipMemcpy(d_R,R,MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex),hipMemcpyDeviceToDevice);
//		error=hipMemcpy(d_psymbolconstellation, psymbolconstellation, M*sizeof(hipComplex),hipMemcpyHostToDevice);
//		error=hipMemcpy(d_s_sub_index, s_sub_index,rho*pathNum*sizeof(int),hipMemcpyHostToDevice);
//		error=hipMemcpy(d_list, list, Nt*(sizeof(int)),hipMemcpyHostToDevice);
	 int sharedMem;
    sharedMem=1*sizeof(hipComplex);
   float duration;
   hipComplex *s_potential1,*s_potential2,*s_potential3,*s_potential4;
   hipComplex *d_s_potential1,*d_s_potential2,*d_s_potential3,*d_s_potential4;
	float *Eu1,*Eu2,*Eu3,*Eu4;
	float *d_Eu1,*d_Eu2,*d_Eu3,*d_Eu4;
	int *sub_index1,*sub_index2,*sub_index3,*sub_index4;
	int *d_sub_index1,*d_sub_index2,*d_sub_index3,*d_sub_index4;
    hipHostAlloc((void**) &s_potential1,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex), hipHostMallocDefault);
    hipHostAlloc((void**) &s_potential2,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex), hipHostMallocDefault);
    hipHostAlloc((void**) &s_potential3,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex), hipHostMallocDefault);
    hipHostAlloc((void**) &s_potential4,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex), hipHostMallocDefault);
    hipHostAlloc((void**) &Eu1,(pathNum/4)*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**) &Eu2,(pathNum/4)*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**) &Eu3,(pathNum/4)*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**) &Eu4,(pathNum/4)*sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void**) &sub_index1,(pathNum/4)*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**) &sub_index2,(pathNum/4)*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**) &sub_index3,(pathNum/4)*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**) &sub_index4,(pathNum/4)*sizeof(int), hipHostMallocDefault);
    hipMalloc((void**) &d_Eu1,(pathNum/4)*sizeof(float));
    hipMalloc((void**) &d_Eu2,(pathNum/4)*sizeof(float));
    hipMalloc((void**) &d_Eu3,(pathNum/4)*sizeof(float));
    hipMalloc((void**) &d_Eu4,(pathNum/4)*sizeof(float));
    hipMalloc((void**) &d_s_potential1,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex));
    hipMalloc((void**) &d_s_potential2,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex));
    hipMalloc((void**) &d_s_potential3,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex));
    hipMalloc((void**) &d_s_potential4,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex));
    hipMalloc((void**) &d_sub_index1,rho*(pathNum/4)*sizeof(int));
    hipMalloc((void**) &d_sub_index2,rho*(pathNum/4)*sizeof(int));
    hipMalloc((void**) &d_sub_index3,rho*(pathNum/4)*sizeof(int));
    hipMalloc((void**) &d_sub_index4,rho*(pathNum/4)*sizeof(int));
    for(count1=0;count1<int(pathNum/4);count1++)
    {
    	for( count2=0;count2<rho;count2++)
    	{
         sub_index1[IDC2D(count2,count1,pathNum/4)]= s_sub_index[IDC2D(count2,count1,pathNum)];
    	}
    }
    for(count1=0;count1<int(pathNum/4);count1++)
    {
    	for(count2=0;count2<rho;count2++)
    	{
         sub_index2[IDC2D(count2,count1,pathNum/4)]= s_sub_index[IDC2D(count2,count1+int(pathNum/4),pathNum)];
    	}
    }
    for(count1=0;count1<int(pathNum/4);count1++)
    {
    	for(count2=0;count2<rho;count2++)
    	{
         sub_index3[IDC2D(count2,count1,pathNum/4)]= s_sub_index[IDC2D(count2,count1+2*int(pathNum/4),pathNum)];
    	}
    }
    for(count1=0;count1<int(pathNum/4);count1++)
    {
    	for(count2=0;count2<rho;count2++)
    	{
         sub_index4[IDC2D(count2,count1,pathNum/4)]= s_sub_index[IDC2D(count2,count1+3*int(pathNum/4),pathNum)];
    	}
    }

    hipStream_t stream1,stream2,stream3,stream4;
    hipStreamCreate(&stream1);hipStreamCreate(&stream2);hipStreamCreate(&stream3);hipStreamCreate(&stream4);
   for(count1=0;count1<stride;count1++)
   {
	hipMemcpyAsync(d_sub_index1,sub_index1,rho*(pathNum/4)*sizeof(int),hipMemcpyHostToDevice,stream1);
	FEpath<<<blockNum, threadNum,sharedMem,stream1>>>(d_s_potential1,d_sub_index1,rho,pathNum, d_Eu1, M,count1);
	error=hipDeviceSynchronize();
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpyAsync(s_potential1,d_s_potential1,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex),hipMemcpyDeviceToHost,stream1);
	hipMemcpyAsync(Eu1,d_Eu1,(pathNum/4)*sizeof(float),hipMemcpyDeviceToHost,stream1);

	hipMemcpyAsync(d_sub_index2,sub_index2,rho*(pathNum/4)*sizeof(int),hipMemcpyHostToDevice,stream2);
	FEpath<<<blockNum, threadNum,sharedMem,stream2>>>(d_s_potential2,d_sub_index2,rho,pathNum, d_Eu2, M,count1);
	error=hipDeviceSynchronize();
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpyAsync(s_potential2,d_s_potential2,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex),hipMemcpyDeviceToHost,stream2);
	hipMemcpyAsync(Eu2,d_Eu2,(pathNum/4)*sizeof(float),hipMemcpyDeviceToHost,stream2);

	hipMemcpyAsync(d_sub_index3,sub_index3,rho*(pathNum/4)*sizeof(int),hipMemcpyHostToDevice,stream3);
	FEpath<<<blockNum, threadNum,sharedMem,stream3>>>(d_s_potential3,d_sub_index3,rho,pathNum, d_Eu3, M,count1);
	error=hipDeviceSynchronize();
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpyAsync(s_potential3,d_s_potential3,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex),hipMemcpyDeviceToHost,stream3);
	hipMemcpyAsync(Eu3,d_Eu3,(pathNum/4)*sizeof(float),hipMemcpyDeviceToHost,stream3);

	hipMemcpyAsync(d_sub_index4,sub_index4,rho*(pathNum/4)*sizeof(int),hipMemcpyHostToDevice,stream4);
	FEpath<<<blockNum, threadNum,sharedMem,stream4>>>(d_s_potential4,d_sub_index4,rho,pathNum, d_Eu4, M,count1);
	hipMemcpyAsync(s_potential4,d_s_potential4,MATRIX_SIZE*(pathNum/4)*sizeof(hipComplex),hipMemcpyDeviceToHost,stream4);
	hipMemcpyAsync(Eu4,d_Eu4,(pathNum/4)*sizeof(float),hipMemcpyDeviceToHost,stream4);
	error=hipDeviceSynchronize();
	printf("%s\n",hipGetErrorString(hipGetLastError()));
   }
//	error=hipMemcpy(s_potential_matrix,d_s_potential_matrix, Nt*sizeof(hipComplex)*pathNum,hipMemcpyDeviceToHost);
	end=clock();
//	hipProfilerStop();
	duration=double(end-start);

	printf("hey %0.4f ", duration);
	printf("\n");
//    memcpy(Eu+0,Eu1,pathNum/4*sizeof(float));
//    memcpy(Eu+pathNum/4,Eu2,pathNum/4*sizeof(float));
//    memcpy(Eu+pathNum/2,Eu3,pathNum/4*sizeof(float));
//    memcpy(Eu+(pathNum*3)/4,Eu4,pathNum/4*sizeof(float));

//	printf("Eu_num is %d", Eu_num);
//    error=hipMemcpy(s_potential_matrix,d_s_potential_matrix,pathNum*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);

//    printf("all the potential symbol vector is:\n");
//    for(count1=0;count1<pathNum;count1++)
//    {
//    	for(int count2=0;count2<Nt;count2++)
//    	{
//    		printf("%0.4f%+0.4fi ", s_potential_matrix[IDC2D(count1,count2,Nt)].x,s_potential_matrix[IDC2D(count1,count2,Nt)].y);
//    	}
//    	printf("\n");
//    }
//    error=hipMemcpy(R,d_R,Nt*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
//    printf("the test upper triangular matrix in kernel is:\n");
//    for(count1=0;count1<Nt;count1++)
//       {
//       	for(int count2=0;count2<Nt;count2++)
//       	{
//       		printf("%0.4f%+0.4fi ", R[IDC2D(count1,count2,Nt)].x,R[IDC2D(count1,count2,Nt)].y);
//       	}
//       	printf("\n");
//       }

//    error=hipMemcpy(Eu,d_Eu,pathNum*sizeof(float),hipMemcpyDeviceToHost);
//    if(error!=hipSuccess)
//    {
////   	printf("Eu returned error code %d, line %d\n", error, __LINE__);
////    			 	exit(EXIT_FAILURE);
//   	}


 //fine out the symbol vector index among all the block output Euclidean distance
//    int Eu_mini_index=0;
//    float Eu_mini_value=Eu[0];
//    for(count1=0;count1<pathNum;count1++)
//    {
//      if(Eu[count1]<Eu_mini_value)
//      {
//    	  Eu_mini_value=Eu[count1];
//    	  Eu_mini_index=count1;
//      }
//    }
    float *d_Eu;
    hipMalloc((void**) &d_Eu,pathNum*sizeof(float));
    hipMemcpy(d_Eu,Eu,pathNum*sizeof(float),hipMemcpyHostToDevice);
    int *Eu_mini_index=(int*)malloc(sizeof(int));
    hipblasIsamin(handle,pathNum,d_Eu,1,Eu_mini_index);
    if(*Eu_mini_index-1<pathNum/4)
    {
    for(count1=0;count1<Nt;count1++)
    {
     s_kernel[list[count1]-1]=s_potential1[IDC2D((MATRIX_SIZE-count1-1),(*Eu_mini_index-1),pathNum/4)];
    }
    }
    else if(*Eu_mini_index-1<pathNum/2)
    {
        for(count1=0;count1<Nt;count1++)
        {
         s_kernel[list[count1]-1]=s_potential2[IDC2D((MATRIX_SIZE-count1-1),(*Eu_mini_index-1-pathNum/4),pathNum/4)];
        }
    }
    else if(*Eu_mini_index-1<(pathNum*3)/4)
    {
        for(count1=0;count1<Nt;count1++)
        {
         s_kernel[list[count1]-1]=s_potential3[IDC2D((MATRIX_SIZE-count1-1),(*Eu_mini_index-1-pathNum/2),pathNum/4)];
        }
    }
    else
    {
        for(count1=0;count1<Nt;count1++)
        {
         s_kernel[list[count1]-1]=s_potential3[IDC2D((MATRIX_SIZE-count1-1),(*Eu_mini_index-1-(pathNum*3)/4),pathNum/4)];
        }
    }

//    	for(int count2=0;count2<MATRIX_SIZE;count2++)
//    	{
//    		printf("%0.4f%+0.4f ", s_share_matrix[IDC2D(count1,count2,MATRIX_SIZE)].x, s_share_matrix[IDC2D(count1,count2,MATRIX_SIZE)].y);
//    	}
//    	printf("\n");
//    }
//       error=hipMemcpy(s_hat,d_s_hat,Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
          if(error!=hipSuccess)
          			   	{
//          			  	printf("s_hat returned error code %d, line %d\n", error, __LINE__);
          //			 	exit(EXIT_FAILURE);
          			   	}
//          for(count1=0;count1<pathNum;count1++)
//          {
//          printf("the unconstrained estimation is:\n");
//          	for(int count2=0;count2<MATRIX_SIZE;count2++)
//          	{
//          		printf("%0.4f%+0.4fi ", s_hat[count2].x, s_hat[count2].y);
//          	}
//          	printf("\n");
//          }

				ret=hipblasDestroy(handle);
			    if (ret != HIPBLAS_STATUS_SUCCESS)
			    {
//			        printf("hipblasDestroy returned error code %d, line(%d)\n", ret, __LINE__);
//			        exit(EXIT_FAILURE);
			    }
			    printf("the s_kernel is :\n");
			    for(count1=0;count1<Nt;count1++)
			    {
			    	printf("%0.4f%+0.4fi ", s_kernel[count1].x, s_kernel[count1].y);
			    }
			    printf("\n");

			   	free(s_sub_index);
//			   	hipFree(d_s_sub_index);
			   	hipFree(d_list);
			   	free(Eu);
			   	hipFree(d_Eu);
//			   	free(s_potential_matrix);
//			   	hipFree(d_s_potential_matrix);
			   	hipFree(d_psymbolconstellation);
			   	free(Eu_mini_index);
//			   	hipFree(d_R);

//			    hipComplex *s_potential1,*s_potential2,*s_potential3,*s_potential4;
//			    hipComplex *d_s_potential1,*d_s_potential2,*d_s_potential3,*d_s_potential4;
//			 	float *Eu1,*Eu2,*Eu3,*Eu4;
//			 	float *d_Eu1,*d_Eu2,*d_Eu3,*d_Eu4;
//			 	int *sub_index1,*sub_index2,*sub_index3,*sub_index4;
//			 	int *d_sub_index1,*d_sub_index2,*d_sub_index3,*d_sub_index4;
			 	hipHostFree(s_potential1);hipHostFree(s_potential2);hipHostFree(s_potential3);hipHostFree(s_potential4);
			 	hipFree(d_s_potential1);hipFree(d_s_potential2);hipFree(d_s_potential3);hipFree(d_s_potential4);
			 	hipHostFree(Eu1);hipHostFree(Eu2);hipHostFree(Eu3);hipHostFree(Eu4);
			 	hipFree(d_Eu1);hipFree(d_Eu2);hipFree(d_Eu3);hipFree(d_Eu4);
			 	hipHostFree(sub_index1);hipHostFree(sub_index2);hipHostFree(sub_index3);hipHostFree(sub_index4);
			 	hipFree(d_sub_index1);hipFree(d_sub_index2);hipFree(d_sub_index3);hipFree(d_sub_index4);
}

