#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
/*
 * this function implement fixed complexity sphere decoding
 * INPUT:
 * y: received signal
 * H: permuted propagation matrix
 * M: modulation scheme, (2: BPSK 4: QPSK, 16: 16QAM, 64: 64QAM)
 * psymbolconstellation: the symbol constellation
 * OUTPUT:
 * s: detection result
 * Eu: Euclidean distance
 */
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_complex.h>
#include <string.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<time.h>
#include"cu_complex_operation.cuh"
#include"common.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<cudaProfiler.h>
#include<hip/hip_runtime_api.h>
#include<hip/device_functions.h>
#define blockNum 4
#define stride 1
__global__ void FEpath(
		hipComplex *R,  //upper triangular matrix after cholesky factorization
		hipComplex *s_hat,  //unconstrained estimation of transmitted symbol vector s
//		hipComplex *s_matrix_share,
		hipComplex *s_potential_matrix,   //the matrix use to store all the solution candidates from all the blocks
		int *s_sub_index,   //full factorial index matrix
//		hipComplex *s,  //decoding results
		float *Eu,  //Euclidean distance
		int pitch_R,    //the number of transmit antennas
		int pitch_index,    //the number of receive antennas
		int pitch_p,
		int M,    //modulation scheme
		int threadNum,    //number of threads
		int *list,     //the permutation list
		hipComplex *psymbolconstellation,
		int index





		)
{
	//need to consider the resource allocation
	int tx=blockIdx.x*blockDim.x+threadIdx.x;     //if the path number is small we can allocate the kernel into one block so that we can use the shared memory
int tid=threadIdx.x;
int Nt=MATRIX_SIZE;
//int bid=blockIdx.x;
//allocate shared memory
	extern __shared__ hipComplex array[];
//	extern __shared__ hipComplex array2[];
//	  extern __shared__ float Eu_vector[];
//	 extern __shared__ float Euclidean[];
//    __shared__ int mini_Eu_index_temp;
//    __shared__ int mini_Eu_index;
//    extern __shared__ int s_sub_index[];
	error_t error;
	int count1, count2,count3,count4;
	__shared__ float d;    //the minimum distance unit between the signal constellation, the distance is usually 2d
	int rho=ceil(sqrt(float(Nt))-1);
	int pathNum=pow(float(M),float(rho));


//	int blockNum=pathNum/(threadNum);
//	int threadNum=1024;free(R_Eu_share);

//	float *distance;
//	int *resu
//#if __CUDA_ARCH__ >+300

//#endif
	   __shared__ hipComplex alpha, beta;
	   alpha.x=1;
	   alpha.y=0;
	   beta.x=0;
	   beta.y=0;

//    float *Eu_vector=Euclidean;
	hipComplex *R_share=array+Nt+2*threadNum;   //the upper trian
//	hipComplex *s_matrix_share=array+Nt*Nt; //the full expansion detection different for different path
//	hipComplex *s_sub_share=array+Nt*(Nt+1)/2;
//	hipComplex *s_sub_share=(hipComplex*)malloc(sizeof(hipComplex))
//    hipComplex *s_hat_share=array+Nt*(Nt+1)/2;
	hipComplex *s_hat_share=array;
    hipComplex *s_temp=array+Nt;
//    hipComplex s_temp[tid];
//    hipComplex *R_Eu_share=array+Nt*(Nt+1)/2+Nt;
    hipComplex *R_Eu_share=(hipComplex*)malloc(Nt*sizeof(hipComplex));
    hipComplex *Eu_norm_share=array+Nt+threadNum;
//    hipComplex Eu_norm_share;
//   float  *Eu_vector=(float*)malloc(threadNum*sizeof(float));
//    hipComplex *s_mini_share=array+Nr*Nt+Nt*threadNum+rho*threadNum+Nt+threadNum+Nt*threadNum+threadNum;


//	   __syncthreads();
    //single expansion
    //
//    for(int i=0;i<pathNum;i+=threadNum)
//    {
//    if((threadNum*blockNum*index+tx)<pathNum)
//    {
   if(tid==0)
   {
    for(count1=0;count1<MATRIX_SIZE;count1++)
    {

		for(count2=0;count2<MATRIX_SIZE;count2++)
		{
//			R_share[(2*MATRIX_SIZE-tid+1)*(tid)/2+count2]=R[IDC2D((count2+tid),tid,pitch_R)];
			R_share[IDC2D(count2,count1,pitch_R)]=R[IDC2D(count2,count1,pitch_R)];
		}


		s_hat_share[count1]=s_hat[count1];

//		printf("the s_hat_share is unchanged? %d, %0.4f%+0.4fi\n", tid, s_hat_share[tid]);
    }
   }
	__syncthreads();
//
//	if(tx==0)
//	{
//		printf("the original upper triangular matrix before decoding is:\n");
//		  for(count1=0;count1<MATRIX_SIZE;count1++)
//		  {
//			  for(count2=0;count2<MATRIX_SIZE;count2++)
//			  {
//				  printf("%0.4f%+0.4fi ", R[IDC2D(count1,count2,pitch_R)].x,R[IDC2D(count1,count2,pitch_R)].y);
//
//			  }
//			  printf("\n");
//		  }
//
//			printf("the shared upper triangular matrix before decoding is:\n");
//			  for(count1=0;count1<MATRIX_SIZE;count1++)
//			  {
//				  for(count2=0;count2<MATRIX_SIZE;count2++)
//				  {
//					  printf("%0.4f%+0.4fi ", R_share[IDC2D(count1,count2,pitch_R)].x,R_share[IDC2D(count1,count2,pitch_R)].y);
//
//				  }
//				  printf("\n");
//			  }
////			  printf("the sub_index is\n");
//////				printf("the shared upper triangular matrix before decoding is:\n");
////				  for(count1=pathNum-16;count1<pathNum;count1++)
////				  {
////					  for(count2=0;count2<rho;count2++)
////					  {
////						  printf("%d ", s_sub_index[IDC2D(count2,count1,pitch_index)]);
////
////					  }
////					  printf("\n");
////				  }
//	}
//		  printf("s_hat is!!!!\n");
//		  for(count1=0;count1<MATRIX_SIZE;count1++)
//		  {
//			  printf("%0.4f%+0.4fi ",s_hat_share[count1].x,s_hat_share[count1].y );
//		  }
//		  printf("kernel is working\n");
//	}
//	for(count4=0;count4<5;count4++)
//	{
//	for(count1=0;count1<rho;count1++)
//	{
//	s_sub_share[IDC2D(tid,count1,rho)]=psymbolconstellation[s_sub_index[IDC2D((index*blockNum*threadNum+tx),count1,rho)]];//read by column major so that it can be continue
//	}
//	if(tx==0)
//	{
//	printf("s_hat_share is\n");
////	for(count2=0;count2<blockDim.x;count2++)
////	{
//	for(count1=0;count1<MATRIX_SIZE;count1++)
//	{
//	printf("%0.4f%+0.4fi ", s_hat_share[count1].x,s_hat_share[count1].y);
//	}
//	printf("\n");
////	}
//	}
//Eu_vector[tid]=0;
	Eu[blockNum*threadNum*index+tx]=0;
for (count1=Nt-1; count1>=0; count1--)
{

		if (count1<Nt-rho)
		{
			s_temp[tid]=s_hat_share[count1];
//			if(tx==0)
//			{
//				printf("the s_temp[tid] is not wrong!!%0.4f%+0.4fi: ", s_hat_share[count1].x,s_hat_share[count1].y);
//				printf("\n");
//			}
			for (count2=count1+1;count2<Nt; count2++)
			{
				s_temp[tid]=complex_add(s_temp[tid],complex_mulcom(complex_div(R_share[IDC2D(count2,count1,pitch_R)],R_share[IDC2D(count1,count1,pitch_R)]),(complex_sub(s_hat_share[count2],s_potential_matrix[IDC2D(count2,(index*blockNum*threadNum+tx),pitch_p)]))));
			}
//			if(tx==0)
//			{
//
//				printf("now the s_temp[tid] is %0.4f%+0.4fi: ", s_temp[tid].x,s_temp[tid].y);
//			}
//			s_share[IDC2D(tid,count1,Nt)]=R_temp[tid];

	//mapping the estimation s_share to the correlated signal constellation(to be continued)
	if(M==2)   //BPSK
	{
//     for(count1=0;count1<Nt-rho; count1++)
//     {
    	 d=sqrt(float(float(1)/float(Nt)));
//    	 s_share_Q[IDC2D(threadIdx.x,count1,Nt)].x=float(int(R_temp[threadIdx.x].x/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
//    	 s_share_Q[IDC2D(threadIdx.x,count1,Nt)].y=0;
//     }0.6565-1.0724i 0.1581+0.1581i 0.3595-0.7887i 0.1581-0.4743i

    	 if(s_temp[tid].x>0)
    	 {
    		 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=d;
    		 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=0;
    	 }
    	 else
    	 {
    		 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=(-d);
    		 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=0;
    	 }
	}
	else if(M==4)   //QPSK
	{
//		if(tx==0)//		printf("the up triangular matrix is:\n");
		//		  for(count1=0;count1<MATRIX_SIZE;count1++)
		//		  {
		//			  for(count2=0;count2<MATRIX_SIZE;count2++)
		//			  {
		//				  printf("%0.4f%+0.4fi ", R_share[IDC2D(count1,count2,MATRIX_SIZE)].x,R_share[IDC2D(count1,count2,MATRIX_SIZE)].y);
		//
		//			  }
		//			  printf("\n");
		//		  }
		//		  printf("kernel is working\n");
//		{
//       #if __CUDA_ARCH__ >=300
		int *result=(int*)malloc(sizeof(int));

		memset(result,0,sizeof(int));
		float *distance=(float*)malloc(M*sizeof(float));
		memset(distance,0,M*sizeof(int));
	    	d=sqrt(float(float(1)/float(Nt)));
           for(count2=0;count2<M;count2++)
           {
        	   switch(count2)
        	   {
        	   case 0:
        	   distance[count2]=sqrt(powf(s_temp[tid].x-(-d),2)+pow(s_temp[tid].y-0,2)); break;
        	   case 1:
        	   distance[count2]=sqrt(powf(s_temp[tid].x-0,2)+pow(s_temp[tid].y-(-d),2)); break;
        	   case 2:
        	   distance[count2]=sqrt(pow(s_temp[tid].x-(d),2)+pow(s_temp[tid].y-0,2)); break;
        	   case 3:
        	   distance[count2]=sqrt(pow(s_temp[tid].x-0,2)+pow(s_temp[tid].y-d,2)); break;
               default:
                   #if __CUDA_ARCH__ >=300
            	   printf("result error code %d\n", error);
                   #endif
            	   break;
        	   }
           }
//#if __CUDA_ARCH__ >=300
//           ret=hipblasIsamin(handle,4,distance,1,result);
//#endif
           float  mini_distance;
  	     int mini_index;
  	     mini_distance=distance[0];
  	     mini_index=1;
  	     for(count3=0;count3<M;count3++)
  	     {
  	    	if(distance[count3]<mini_distance)
  	    	{
  	    		mini_distance=distance[count3];
  	    		mini_index=count3+1;
  	    	}
  	     }



//           {
//             #if __CUDA_ARCH__ >=300
//        	   printf("hipblasIsamin failed return error code %d, line %d\n",error,__LINE__);
////        	   exit(EXIT_bbbb
//#endif
//           }

           switch (mini_index)
           {
           case 1:
        	s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=-d;
        	s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=0;
        	break;
           case 2:
           	s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=0;
           	s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=-d;
           	break;
           case 3:
           	s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=d;
           	s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=0;
           	break;
           case 4:
       	  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=0;
       	  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=d;
       	  break;
           default:
               #if __CUDA_ARCH__ >=300
        	   printf("result error code %d\n", error);
               #endif
           }
           free(distance);
           free(result);
	}
//#if __CUDA_ARCH__ >=300
//          hipblasDestroy(handle);
//#endif
	else if(M==16)  //16QAM
	{
//	     for(count1=0;count1<Nt-rho; count1++)
//	     {
	    	 d=sqrt(float(3)/(2* (float)(Nt*(M-1))));
//	    	 if(tx==0)
//	    	 {
//	    	 printf("the distance of constellation is: %0.4f ", d);
//	    	 printf("the s_temp[tid] is %0.4f%+0.4fi ", s_temp[tid].x,s_temp[tid].y);
//	    	 }
//	  s_share_Q[IDC2D(tid,count1,Nt)].x=float(int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
//	  s_share_Q[IDC2D(threadIdx.x,count1,Nt)].y=float(int(s_share[IDC2D(threadIdx.x,count1,Nt)].y/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
//	     if(s_temp[tid].x<(-2*d))
//	    	{
//	    	  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=(-3*d);
//	    	}
//	    else if(s_temp[tid].x>(2*d))
//	    	{
//	    	  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=(3*d);
//	    	}
//	    else if(s_temp[tid].x>=0&&s_temp[tid].x<=2*d)
//	    	{
//	    	  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=d;
//	    	}
//	    else if(s_temp[tid].x>=(-2*d)&&s_temp[tid].x<=0)
//	        {
//	    	  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=(-d);
//	    	}
//
//
//
//
//	    if(s_temp[tid].y<(-2*d))
//	    	{
//	    	   s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=(-3*d);
//	    	}
//	   else if(s_temp[tid].y>(2*d))
//	    	 {
//	    	  s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=(3*d);
//	         }
//	   else if(s_temp[tid].y>=0&&s_temp[tid].y<=(2*d))
//	        {
//	    	s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=d;
//	    	}
//	    else if(s_temp[tid].y>=(-2*d)&&s_temp[tid].y<=0)
//	       {
//	        s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=(-d);
//	       }
//	     }

	    	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].x=2*d*int(s_temp[tid].x/(2*d))+d*(int(s_temp[tid].y/sqrt(pow(s_temp[tid].x,2)+pow(s_temp[tid].x,2))));
	    	 s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)].y=2*d*int(s_temp[tid].y/(2*d))+d*(int(s_temp[tid].y/sqrt(pow(s_temp[tid].y,2)+pow(s_temp[tid].y,2))));
	}
	else if(M==64)   //64QAM
		{
//	     for(count1=0;count1<Nt-rho; count1++)
//	     {
	    	 d=sqrt(3/(2* (float)(Nt*(M-1))));
//	     s_potential_matrix[IDC2D(tid,count1,Nt)].x=float(int(s_potential_matrix[IDC2D(threadIdx.x,count1,Nt)].x/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
//	    s_matrix_Q[IDC2D(threadIdx.x,count1,Nt)].y=float(int(s_potential_matrix[IDC2D(threadIdx.x,count1,Nt)].y/d)*d)+float(int(round(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d-int(s_share[IDC2D(threadIdx.x,count1,Nt)].x/d)))*d);
	     }
//		}
		}
	else
	{
//			s_share[IDC2D(tid,count1,Nt)]=s_sub_share[IDC2D(tx,(Nt-count1-1),Nt)];
//		s_potential_matrix[IDC2D(count2,(index*blockNum*threadNum+tx),pitch_p)]=s_sub_share[IDC2D(tid,(Nt-count1-1),rho)];
		s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)]=psymbolconstellation[s_sub_index[IDC2D((Nt-count1-1),(index*blockNum*threadNum+tx),pitch_index)]];
//		printf("the s_potential_matrix in each thread is:\n");
//		printf("%0.4f%+0.4fi ", s_potential_matrix[IDC2D(tid,count1,Nt)].x,s_potential_matrix[IDC2D(tid,count1,Nt)].y);
//		printf("\n");
	}
//		if(tid==0)
//		{
//
//			printf("the decoding matrix is:%0.4f%+0.4fi ",s_potential_matrix[IDC2D(count2,(index*blockNum*threadNum+tx),pitch_p)] );
//		}

//		R_Eu_share[IDC2D(tid,count1,Nt)]=complex_sub(s_potential_matrix[IDC2D(count2,(index*blockNum*threadNum+tx),pitch_p)],s_hat_share[count1]);
		R_Eu_share[count1]=complex_sub(s_potential_matrix[IDC2D(count1,(index*blockNum*threadNum+tx),pitch_p)],s_hat_share[count1]);
		Eu_norm_share[tid]=beta;
		for(count3=count1;count3<MATRIX_SIZE;count3++)
		{
		Eu_norm_share[tid]=complex_add(Eu_norm_share[tid],complex_mulcom(R_share[IDC2D(count3,count1,pitch_R)],R_Eu_share[count3]));
		}
//		Eu_vector[tid]=Eu_vector[tid]+pow(Eu_norm_share.x,2)+pow(Eu_norm_share.y,2);
		Eu[(index*blockNum*threadNum+tx)]=Eu[(index*blockNum*threadNum+tx)]+pow(Eu_norm_share[tid].x,2)+pow(Eu_norm_share[tid].y,2);

	}
free(R_Eu_share);
//free(Eu_vector);
__syncthreads();
//if(tid>=0&&tid<threadNum)
//{
//	Eu[(index*blockNum*threadNum+tx)]=Eu_vector[tid];
//}
//__syncthreads();
//    }

}

//host
void FCSD_decoding(
		hipComplex *R,  //upper triangular matrix after cholesky factorization store in device side
//		hipComplex *s_sub, //the sub brute force rho vector matrix
		hipComplex *d_s_hat,  //unconstrained estimation of transmitted symbol vector s
		hipComplex *s_kernel,  //quantization of estimation ,decoding results
//		hipComplex *Eu,  //Euclidean distance
		int Nt,    //the number of transmit antennas
		int Nr,    //the number of receive antennas
		int M,    //modulation scheme
		int *list,   //the permutation list
		hipComplex *psymbolconstellation //the symbol constellation
		)
{
//brute force search determine the vector results of the full expansion
	int rho=ceil(sqrt(Nt)-1);
//	hipComplex *ss;
//	ss=(hipComplex*)malloc(MATRIX_SIZE*sizeof(hipComplex));
//	hipComplex *s_sub;
//	s_sub=(hipComplex*)malloc(pow(M,rho)*rho*sizeof(hipComplex));   //all the possible full expansion sub vector
	int  pathNum;
	pathNum=pow(M,rho);
	int *d_list,*d_s_sub_index;
	int *s_sub_index=(int*)calloc(1,rho*pow(M,rho)*sizeof(int));
	fullfact(rho,M,s_sub_index);    //get  the indexes of all the possible rho length symbol vectors
//	int blockNum=BLOCK_NUM;   //determined by the path number
//	int pathNum=pow(M,rho);  //number of search path
	int threadNum=ceil(pathNum/(blockNum*stride)); //determined by the path number
	float *Eu,*d_Eu;
	Eu=(float*)calloc(1,blockNum*sizeof(float));
	hipComplex *s_potential_matrix=(hipComplex*)calloc(1,pathNum*Nt*sizeof(hipComplex));
	hipComplex  *d_R, *d_s_potential_matrix,*d_psymbolconstellation;
	int count1;
//	int *j;
//	j=(int*)malloc(sizeof(int));
	hipblasHandle_t handle;
		hipblasStatus_t ret;
		hipError_t error;
		size_t pitch_R,pitch_potential,pitch_index;
		ret=hipblasCreate(&handle);
	    error=hipMallocPitch((void**) &d_R, &pitch_R, MATRIX_SIZE*sizeof(hipComplex),MATRIX_SIZE);
	    error=hipMallocPitch((void**) &d_s_sub_index, &pitch_index,pathNum*sizeof(int),rho);
		error=hipMallocPitch((void**) &d_s_potential_matrix,&pitch_potential, pathNum*sizeof(hipComplex),Nt);
		error=hipMalloc((void**) &d_list, MATRIX_SIZE*sizeof(int));
		error=hipMalloc((void**) &d_Eu, pathNum*sizeof(float));
        error=hipMalloc((void**) &d_psymbolconstellation, M*sizeof(hipComplex));
        error=hipMemcpy2D(d_R,pitch_R,R,sizeof(hipComplex),sizeof(hipComplex),MATRIX_SIZE*MATRIX_SIZE,hipMemcpyDeviceToDevice);
		error=hipMemcpy(d_psymbolconstellation, psymbolconstellation, M*sizeof(hipComplex),hipMemcpyHostToDevice);
		error=hipMemcpy2D(d_s_sub_index,pitch_index, s_sub_index,pathNum*sizeof(int), int(pow(M,rho))*sizeof(int),rho,hipMemcpyHostToDevice);
		error=hipMemcpy(d_list, list, Nt*(sizeof(int)),hipMemcpyHostToDevice);
	 int sharedMem;
    sharedMem=6000*sizeof(hipComplex);
   float duration;
   clock_t start, end;
   start=clock();
   for(count1=0;count1<stride;count1++)
   {
	FEpath<<<blockNum, threadNum,sharedMem>>>(d_R, d_s_hat, d_s_potential_matrix,d_s_sub_index, d_Eu, int(pitch_R/sizeof(hipComplex)), int(pitch_index/sizeof(int)),int (pitch_potential/sizeof(hipComplex)), M,threadNum,d_list,d_psymbolconstellation,count1);
	error=hipDeviceSynchronize();
	printf("%s\n",hipGetErrorString(hipGetLastError()));
   }
	end=clock();
//	hipProfilerStop();
	duration=double(end-start);
	printf("hey %0.4f ", duration);
	printf("\n");

		if(error!=hipSuccess)
		{
//		printf("hipDeviceSynchronize returned error code %d, line %d\n", error, __LINE__);
//				 	exit(EXIT_FAILURE);
		}

//	printf("Eu_num is %d", Eu_num);
//    error=hipMemcpy(s_potential_matrix,d_s_potential_matrix,pathNum*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
error=hipMemcpy2D(s_potential_matrix,pathNum*sizeof(hipComplex), d_s_potential_matrix,pitch_potential, pathNum*sizeof(hipComplex),MATRIX_SIZE,hipMemcpyDeviceToHost);
//    printf("all the potential symbol vector is:\n");
//    for(count1=0;count1<pathNum;count1++)
//    {
//    	for(int count2=0;count2<Nt;count2++)
//    	{
//    		printf("%0.4f%+0.4fi ", s_potential_matrix[IDC2D(count1,count2,Nt)].x,s_potential_matrix[IDC2D(count1,count2,Nt)].y);
//    	}
//    	printf("\n");
//    }
//    error=hipMemcpy(R,d_R,Nt*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
//    printf("the test upper triangular matrix in kernel is:\n");
//    for(count1=0;count1<Nt;count1++)
//       {
//       	for(int count2=0;count2<Nt;count2++)
//       	{
//       		printf("%0.4f%+0.4fi ", R[IDC2D(count1,count2,Nt)].x,R[IDC2D(count1,count2,Nt)].y);
//       	}
//       	printf("\n");
//       }

//    error=hipMemcpy(Eu,d_Eu,pathNum*sizeof(float),hipMemcpyDeviceToHost);
//    if(error!=hipSuccess)
//    {
////   	printf("Eu returned error code %d, line %d\n", error, __LINE__);
////    			 	exit(EXIT_FAILURE);
//   	}


 //fine out the symbol vector index among all the block output Euclidean distance
//    int Eu_mini_index=0;
//    float Eu_mini_value=Eu[0];
//    for(count1=0;count1<pathNum;count1++)
//    {
//      if(Eu[count1]<Eu_mini_value)
//      {
//    	  Eu_mini_value=Eu[count1];
//    	  Eu_mini_index=count1;
//      }
//    }
    int *Eu_mini_index=(int*)malloc(sizeof(int));
    hipblasIsamin(handle,pathNum,d_Eu,1,Eu_mini_index);
    for(count1=0;count1<Nt;count1++)
    {
     s_kernel[list[count1]-1]=s_potential_matrix[IDC2D((MATRIX_SIZE-count1-1),(*Eu_mini_index-1),Nt)];
    }

//    	for(int count2=0;count2<MATRIX_SIZE;count2++)
//    	{
//    		printf("%0.4f%+0.4f ", s_share_matrix[IDC2D(count1,count2,MATRIX_SIZE)].x, s_share_matrix[IDC2D(count1,count2,MATRIX_SIZE)].y);
//    	}
//    	printf("\n");
//    }
//       error=hipMemcpy(s_hat,d_s_hat,Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);
          if(error!=hipSuccess)
          			   	{
//          			  	printf("s_hat returned error code %d, line %d\n", error, __LINE__);
          //			 	exit(EXIT_FAILURE);
          			   	}
//          for(count1=0;count1<pathNum;count1++)
//          {
//          printf("the unconstrained estimation is:\n");
//          	for(int count2=0;count2<MATRIX_SIZE;count2++)
//          	{
//          		printf("%0.4f%+0.4fi ", s_hat[count2].x, s_hat[count2].y);
//          	}
//          	printf("\n");
//          }

				ret=hipblasDestroy(handle);
			    if (ret != HIPBLAS_STATUS_SUCCESS)
			    {
//			        printf("hipblasDestroy returned error code %d, line(%d)\n", ret, __LINE__);
//			        exit(EXIT_FAILURE);
			    }
			    printf("the s_kernel is :\n");
			    for(count1=0;count1<Nt;count1++)
			    {
			    	printf("%0.4f%+0.4fi ", s_kernel[count1].x, s_kernel[count1].y);
			    }
			    printf("\n");

			   	free(s_sub_index);
			   	hipFree(d_s_sub_index);
			   	hipFree(d_list);
			   	free(Eu);
			   	hipFree(d_Eu);
			   	free(s_potential_matrix);
			   	hipFree(d_s_potential_matrix);
			   	hipFree(d_psymbolconstellation);
			   	free(Eu_mini_index);
//			   	hipFree(d_R);

}

