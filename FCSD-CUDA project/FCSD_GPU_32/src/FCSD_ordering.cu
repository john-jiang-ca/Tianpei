#include "hip/hip_runtime.h"
/*
 * FCSD_ordering.c
 *this function implement the channel ordering of fixed complexity sphere decoding algorithm
 *INPUT:
 *pH: original propagation channel matrix
 *list: permutation list
 *pH_permuted: permuted propagation channel matrix
 *  Created on: Jun 26, 2014
 *      Author: Tianpei Chen
 *      Email: tianpei.chen@mail.mcgill.ca
 */

//#ifndef FCSD_ORDERING_H_
//#define FCSD_ORDERING_H_
#include"common.h"
#include<hipblas.h>
#include<hip/hip_complex.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include <math.h>
#include<assert.h>
#include<string.h>
#include<hip/hip_runtime.h>
void FCSD_ordering(
		hipComplex *pH,
		int *list,
		hipComplex *d_pH_permuted
)
{

	int count, count1,count2,count3;
	int mm,*j,*list_temp;
	j=(int*)malloc(sizeof(int));
	list_temp=(int*)malloc(MATRIX_SIZE*sizeof(int));
//	int *d_j;
//	hipMalloc((void**) &d_j, sizeof(int));
	hipComplex *pH_loop;
	pH_loop=(hipComplex*)malloc(MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
	int N=ceil(sqrt(MATRIX_SIZE)-1);//
	hipblasHandle_t handle;
	hipblasStatus_t ret;
	hipError_t error;
	ret=hipblasCreate(&handle);

    hipComplex *d_pH, *d_Pprod, *d_Pinv;
    hipComplex alpha,beta;
    alpha.x=1;
    alpha.y=0;
    beta.x=0;
    beta.y=0;
	  clock_t start,end;
	  start=clock();
		error=hipMalloc((void**) &d_Pprod, (MATRIX_SIZE)*(MATRIX_SIZE)*sizeof(hipComplex));//store the wishart matrix
			if(error!=hipSuccess)
			{
				printf("hipMalloc d_Pprod returned error code %d, line %d\n", error, __LINE__);
//				exit(EXIT_FAILURE);
			}
			error=hipMalloc((void**) &d_Pinv, (MATRIX_SIZE)*(MATRIX_SIZE)*sizeof(hipComplex));//store the inverse of wishart matrix
			if(error!=hipSuccess)
			{
				printf("hipMalloc d_Pinv returned error code %d, line %d\n", error, __LINE__);
				exit(EXIT_FAILURE);
			}
			hipMalloc((void**) &d_pH,(MATRIX_SIZE)*MATRIX_SIZE*sizeof(hipComplex));



	for(count2=0;count2<MATRIX_SIZE-1;count2++)
	{
			if(count2==0)
			{
			hipMemcpy(d_pH,pH,(MATRIX_SIZE-count2)*MATRIX_SIZE*sizeof(hipComplex),hipMemcpyHostToDevice);
			}
			else
			{
			hipMemcpy(d_pH,pH_loop,(MATRIX_SIZE-count2)*MATRIX_SIZE*sizeof(hipComplex),hipMemcpyHostToDevice);
			}


		ret=hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_C, MATRIX_SIZE-count2, MATRIX_SIZE-count2, MATRIX_SIZE, &alpha, d_pH, MATRIX_SIZE-count2, d_pH, MATRIX_SIZE-count2, &beta, d_Pprod, MATRIX_SIZE-count2);
	    if (ret != HIPBLAS_STATUS_SUCCESS)
	    {
	        printf("hipblasSgemm returned error code %d, line(%d)\n", ret, __LINE__);
	        exit(EXIT_FAILURE);
	    }



	    int column=MATRIX_SIZE-count2;
	    int row=MATRIX_SIZE;
	    double duration;

	    start=clock();
	    MATRIX_INVERSE(d_Pprod,d_Pinv,row,column);
	    end=clock();
	    duration=double((end-start));
	    hipDeviceSynchronize();
	    printf("the duration of matrix inverse GPU is %0.4f:\n", duration);
		if(count2<N)
		{
			hipblasIcamax(handle, (MATRIX_SIZE-count2),d_Pinv,(MATRIX_SIZE-count2),j);
		}
		else
		{
			hipblasIcamin(handle, (MATRIX_SIZE-count2),d_Pinv,(MATRIX_SIZE-count2),j);
		}

		  list_temp[count2]=list[count2+*j-1];   //choose the *j th element in the new list to be detected first
					    list[count2+*j-1]=0;
					for (count3=0;count3<MATRIX_SIZE-count2-1;count3++)
					{//				double durationMalloc;
						//				start=clock();
						//				hipMalloc((void**) &d_pH,MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
						//				hipMemcpy(d_pH,pH,MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex),hipMemcpyHostToDevice);
						////				hipMemcpy(pH,d_pH,MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex),hipMemcpyDeviceToHost);
						//			    end=clock();
						//			    durationMalloc=double(end-start);
						//			    printf("the memcpy time is %0.4f:\n", durationMalloc);



						//		error=hipMalloc((void**) &d_pH, (MATRIX_SIZE-count2)*MATRIX_SIZE*sizeof(hipComplex));// store the Hi after each refresh
						//		if(error!=hipSuccess)
						//		{
						//			printf("hipMalloc d_pH returned error code %d, line %d\n", error, __LINE__);
						//			exit(EXIT_FAILURE);
						//		}
						for(int count4=count2;count4<MATRIX_SIZE;count4++)
						{
							if(list[count4]!=0)
							{
						list_temp[count2+count3+1]=list[count4];   //choose the next MATRIX_SIZE-(count2+1) to form the next sequence, the sequence order is the same
						list[count4]=0;
							break;
							}
						}
					}
					for(count3=0;count3<MATRIX_SIZE;count3++)
					{
					list[count3]=list_temp[count3];
					}

	hipComplex *pH_temp1=(hipComplex*)malloc((MATRIX_SIZE)*(MATRIX_SIZE-count2-1)*sizeof(hipComplex));

						for(count3=0;count3<MATRIX_SIZE;count3++)
						 {
							 for(count1=0;count1<MATRIX_SIZE-count2;count1++)
							 {
								 if(count1<(*j-1))
								 {
			//		              pH_temp1[IDC2D(count3,count1,(MATRIX_SIZE-count2-1))]=pH_loop[IDC2D(count3,count1,(MATRIX_SIZE-count2))];
									 pH_temp1[IDC2D(count3,count1,(MATRIX_SIZE-count2-1))]=pH[IDC2D(count3,count1,(MATRIX_SIZE-count2))];
								 }
								 else if(count1>(*j-1))
								 {
			//						 pH_temp1[IDC2D(count3,count1-1,(MATRIX_SIZE-count2-1))]=pH_loop[IDC2D(count3,count1,(MATRIX_SIZE-count2))];
									 pH_temp1[IDC2D(count3,(count1-1),(MATRIX_SIZE-count2-1))]=pH[IDC2D(count3,count1,(MATRIX_SIZE-count2))];
								 }
							 }
						 }
pH_loop=(hipComplex*)realloc(pH_loop,(MATRIX_SIZE)*(MATRIX_SIZE-count2-1)*sizeof(hipComplex));
						for(count3=0;count3<MATRIX_SIZE;count3++)
						 {
							 for(count1=0;count1<MATRIX_SIZE-count2-1;count1++)
							 {
			//		         pH_loop[IDC2D(count3,count1,(MATRIX_SIZE-count2-1))]= pH_temp1[IDC2D(count3,count1,(MATRIX_SIZE-count2-1))];
								pH_loop[IDC2D(count3,count1,(MATRIX_SIZE-count2-1))]=pH_temp1[IDC2D(count3,count1,(MATRIX_SIZE-count2-1))];
							 }
						 }


free(pH_temp1);
}
end=clock();
double duration=double(end-start);
	hipComplex *pH_permuted;
	pH_permuted=(hipComplex*)malloc(MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
	for(count1=0;count1<MATRIX_SIZE;count1++)
	{
	for(count2=0;count2<MATRIX_SIZE;count2++)
	{
	pH_permuted[IDC2D(count1,(MATRIX_SIZE-count2-1),MATRIX_SIZE)]=pH[IDC2D(count1,list[count2]-1,MATRIX_SIZE)];
	}
	}
  hipMemcpy(d_pH_permuted,pH_permuted,MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex),hipMemcpyHostToDevice);
    ret=hipblasDestroy(handle);
    if (ret != HIPBLAS_STATUS_SUCCESS)
       {
           printf("cublasDestory returned error code %d, line(%d)\n", ret, __LINE__);
           exit(EXIT_FAILURE);
       }

hipFree(d_pH);
hipFree(d_Pprod);
hipFree(d_Pinv);
free(list_temp);
free(j);
free(pH_loop);
free(pH_permuted);

}



