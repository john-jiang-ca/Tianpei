#include "hip/hip_runtime.h"
/*
 * row_column_transformation.cu
 *
 *  Created on: Jul 7, 2014
 *      Author: tianpei.chen@mail.mcgill.ca
 */
#include"common.h"
#include<hip/hip_complex.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include"cu_complex_operation.cuh"



__global__ void MATRIX_ROWCOLUMNT_kernel(hipComplex *Hr,
		hipComplex *Hc, int row, int column)
{
	int i=blockIdx.x;
	int j=threadIdx.x;


     Hc[IDC2D(j,i,row)]=Hr[IDC2D(j,i,column)];
     __syncthreads();

}




__global__ void MATRIX_COLUMNROWT_kernel(hipComplex *Hc,
		hipComplex *Hr, int row, int column)
{
	int i=blockIdx.x;
	int j=threadIdx.x;


     Hr[IDC2D(i,j,column)]=Hc[IDC2D(i,j,row)];
     __syncthreads();


}





void MATRIX_COLUMNROWT(hipComplex *Hc, hipComplex *Hr, int row, int column){

	hipComplex *d_pCC, *d_pCCR;

	hipMalloc((void**)&d_pCCR,row*column*sizeof(hipComplex));
	hipMalloc((void**)&d_pCC,row*column*sizeof(hipComplex));
	hipMemcpy(d_pCC,Hc,row*column*sizeof(hipComplex),hipMemcpyHostToDevice);
	dim3 blockDim(column,1);
	MATRIX_COLUMNROWT_kernel<<<row,column,0>>>(d_pCC, d_pCCR, row, column);
	hipDeviceSynchronize();
	hipMemcpy(Hr,d_pCCR,row*column*sizeof(hipComplex),hipMemcpyDeviceToHost);
	hipFree(d_pCC);
	hipFree(d_pCCR);

}
void MATRIX_ROWCOLUMNT(hipComplex *Hr, hipComplex *Hc, int row,int column){

	hipComplex *d_pCC, *d_pCCR;

	hipMalloc((void**)&d_pCCR,row*column*sizeof(hipComplex));
	hipMalloc((void**)&d_pCC,row*column*sizeof(hipComplex));
	hipMemcpy(d_pCC,Hr,row*column*sizeof(hipComplex),hipMemcpyHostToDevice);
	dim3 blockDim(column,1);
	MATRIX_ROWCOLUMNT_kernel<<<row,column,0>>>(d_pCC, d_pCCR, row,column);
	hipDeviceSynchronize();
	hipMemcpy(Hc,d_pCCR,row*column*sizeof(hipComplex),hipMemcpyDeviceToHost);
	hipFree(d_pCC);
	hipFree(d_pCCR);

}
