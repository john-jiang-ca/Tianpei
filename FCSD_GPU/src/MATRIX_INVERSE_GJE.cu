#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "cu_complex_operation.cuh"
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
//#include<hip/hip_runtime_api.h>
//#include<helper_functions.h>
#include<time.h>
#include<hip/hip_runtime_api.h>
#include"common.h"




//pivot row normal1600ization
__global__ void normalizePivotRow( hipComplex *H, int index, int lda ) {
	int tid=threadIdx.x;
__shared__ hipComplex pivotValue;
if(tid==0)
{
//	printf("check the propagation matrix\n");
//	for(int count1=0;count1<N1;count1++)
//	{
//		for(int count2=0;count2<2*N1;count2++)
//		{
//			printf("%0.4f%+0.4fi ", H[IDC2D(count1,count2,lda)].x, H[IDC2D(count1,count2,lda)].y);
//		}
//		printf("\n");
//	}
	}
 if(tid<lda){

			if ( tid == 0 ) // First thread of each block loads pivotValue
			{
			pivotValue = H[ IDC2D( index, index, lda) ];
			}
			__syncthreads();

//			printf("the pivot value is %0.4f%+0.4fi :\n", pivotValue.x,pivotValue.y);
			H[ IDC2D( index, tid, lda )]=complex_div(H[ IDC2D( index, tid, lda )],pivotValue);
			__syncthreads();
 }

//printf("the thread Id of matrix iverse is:\n");
//printf("%d ", tid);
//printf("the row of the pivot H is:\n");
//printf("%0.4f%+0.4fi ", H[ IDC2D( index, tid, lda )].x,H[ IDC2D( index, tid, lda )].y );


}



//elements update
__global__ void linearMge( hipComplex *matrix, int index, int lda,int BLOCKNUM) {
	int tx=blockIdx.x*blockDim.x+threadIdx.x;
	int bid =threadIdx.x;
	int tid = threadIdx.y;
	 extern __shared__ hipComplex array[ ];
	__shared__ hipComplex zero;
		zero.x=0;
		zero.y=0;
//	extern __shared__ hipComplex matrixPivotValue[];
	hipComplex *matrixPivotValue=array;
	 hipComplex *multColumn=array+blockDim.x;
//			 int(int((lda)/2)/BLOCKNUM);
	 hipComplex *matrixRow=array+blockDim.x+lda;
if(tx<int(lda/2))
{
	if ( tx!=index ) {
		if(tid==0)
		{
	// Each block loads the value of the pivot Row to be substracted
	matrixPivotValue[bid] = matrix[ IDC2D( tx, index, lda )];
//	resultPivotValue = result[ IDC2D( index, x, lda )];
	matrix[ IDC2D(tx, index, lda )]=zero;
//	printf("the zeroing tx is %d:\n", tx);
		}
	}
	else
	{
		matrixPivotValue[bid]=zero;
	}
	__syncthreads();
	if(tid==0)
	{
//	printf("\n");
//	printf("the pivot column is:\n");
//	printf("%0.4f%+0.4fi, %d ",matrix[ IDC2D(tx, index, lda )].x,matrix[ IDC2D(tx, index, lda )].y, tx);
	}
	if(bid==0)
	{
	multColumn[ tid ] = matrix[ IDC2D( index, tid, lda )];
	}

	matrixRow[ IDC2D(bid,tid,lda) ] = matrix[ IDC2D( tx, tid, lda )];
//	resultRow[ ty ] = result[ IDC2D( y, x, lda )];
	__syncthreads();
//	newMatrixValue =matrix[ IDC2D( ty, x, lda )];
	if(tid!=index)
	{
matrix[ IDC2D(tx, tid, lda) ]=complex_sub(matrixRow[IDC2D(bid,tid,lda)],complex_mulcom( multColumn[tid],matrixPivotValue[bid]));
	}
	// Copy to the matrix
//	matrix[ IDC2D( ty, x, lda) ] = newMatrixValue;
	__syncthreads();

//	printf("the update value is:\n");
//		printf("%0.4f%+0.4fi ",matrix[ IDC2D( index, tid, lda )].x,matrix[ IDC2D( index, tid, lda )].y );
//	printf("the index of the whole matrix is %d:\n", tx);
//	printf("the index of the matrix in one block is: %d\n", bid);
}
}
__global__ void transfer(
		hipComplex *d_H,
		hipComplex *R_inv,
		int size
		)
{
	int bid=blockIdx.x*blockDim.x+threadIdx.x;
	int tid=threadIdx.y;
	if(bid>=0&&bid<size&&tid>=0&&tid<size)
	{
	R_inv[IDC2D(bid,tid,size)]=d_H[IDC2D(bid,(tid+size),2*size)];
	}
	__syncthreads();
//	if(bid==0&&tid==0)
//	{
//		printf("the result of transfer is:\n");
//		for(int count1=0;count1<size;count1++)
//		{//#define BLOCKNUM 4
	//Row switching
//			for(int count2=0;count2<size;count2++)
//			{
//				printf("%0.4f%+0.4fi ", R_inv[IDC2D(count1,count2,size)].x,R_inv[IDC2D(count1,count2,size)].y);
//			}
//			printf("\n");
//		}
//	}

}
__global__ void initial
(
		hipComplex *d_H,
		hipComplex *matrix,
		int size
		)
{
	int bid=blockIdx.x*blockDim.x+threadIdx.x;
	int tid=threadIdx.y;
	if(bid<size)
	{
	if(tid<size)
	{
	matrix[IDC2D(bid,tid,2*size)]=d_H[IDC2D(bid,tid,size)];
	}
	else if(tid==bid+size)
	{
		matrix[IDC2D(bid,tid,2*size)].x=1;
		matrix[IDC2D(bid,tid,2*size)].y=0;
	}
	else
	{
		matrix[IDC2D(bid,tid,2*size)].x=0;
		matrix[IDC2D(bid,tid,2*size)].y=0;
	}
	}
	__syncthreads();
//	if(bid==0&&tid==0)
//	{
//		printf("the result of initial is:\n");
//		for(int count1=0;count1<size;count1++)
//		{
//			for(int count2=0;count2<2*size;count2++)
//			{
//				printf("%0.4f%+0.4fi ", matrix[IDC2D(count1,count2,2*size)].x,matrix[IDC2D(count1,count2,2*size)].y);
//			}
//			printf("\n");
//		}
//	}
}
void MATRIX_INVERSE(
	hipComplex *H,  //input square matrix stored in row
	hipComplex *R,   //the inversion of the matrix H stored in row
	int row,        // the number of the rows
	int column      //the number of columns of the H_row
)
{
	int BLOCKNUM=16;
	hipError_t error;
	int count1,count2;
	if(column<=8)
	{
		BLOCKNUM=1;
	}
	dim3 thread1(ceil(float(column)/float(BLOCKNUM)),2*column);
	dim3 thread2(ceil(float(column)/float(BLOCKNUM)),column);
	hipComplex *d_matrix;
	clock_t start, end;
	double duration;
	hipMalloc((void**) &d_matrix, column*2*column*sizeof(hipComplex));
	start=clock();
	initial<<<BLOCKNUM,thread1>>>(H,d_matrix,column);
	end=clock();
	duration=double(end-start);
//	error=hipDeviceSynchronize();
	if(error!=hipSuccess)
	{
	printf("error=%s\n",hipGetErrorString(hipGetLastError()));
	}
	dim3 blockDim(ceil(float(column)/float(BLOCKNUM)),2*column);
	start=clock();
	for(count1=0; count1<column; count1++)
	{
   //shared memory to be changed
		normalizePivotRow<<<1,2*column>>>( d_matrix, count1, 2*column );

//		error=hipDeviceSynchronize();
//        if(error!=hipSuccess)
//        {
		printf("error=%s\n",hipGetErrorString(hipGetLastError()));
//        }
	linearMge<<<BLOCKNUM,blockDim,5000*sizeof(hipComplex)>>>( d_matrix, count1, 2*column,BLOCKNUM );
//	error=hipDeviceSynchronize();
	if(error!=hipSuccess)
//	{
	printf("error=%s\n",hipGetErrorString(hipGetLastError()));
//	}

	}
	end=clock();
	duration=double(end-start);
	start=clock();
  transfer<<<BLOCKNUM,thread2>>>(d_matrix,R,column);
	end=clock();
	duration=double(end-start);
  error=hipDeviceSynchronize();
//	if(error!=hipSuccess)
//	{
  printf("error=%s\n",hipGetErrorString(hipGetLastError()));
//	}
  hipFree(d_matrix);
//	free(R);
//	free(H);

}




