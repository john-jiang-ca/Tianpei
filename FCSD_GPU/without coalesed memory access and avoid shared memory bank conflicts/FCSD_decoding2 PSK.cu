#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
/*
 * this function implement fixed complexity sphere decoding
 * INPUT:
 * y: received signal
 * H: permuted propagation matrix
 * M: modulation scheme, (2: BPSK 4: QPSK, 16: 16QAM, 64: 64QAM)
 * psymbolconstellation: the symbol constellation
 * OUTPUT:
 * s: detection result
 * Eu: Euclidean distance
 */
#include <stdio.h>
#include <stdlib.h>
#include<hip/hip_complex.h>
#include <string.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<time.h>
#include"cu_complex_operation.cuh"
#include"common.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<cudaProfiler.h>
#include<hip/hip_runtime_api.h>
#include<hip/device_functions.h>
#define threadNum 1024
#define blockNum 4
#define stride 1





/*
 * in this version I applied colesced memory accesss to all the vector and matrix, with all the matrix stored in row major different threads reading one column
 * with all the matrix stored in column major, different threads reading one row
 * uitilize consecutive computation power
 */
__constant__ hipComplex d_R[MATRIX_SIZE*MATRIX_SIZE],d_psymbolconstellation[16],d_constant_shat[MATRIX_SIZE];
__constant__ int d_list[MATRIX_SIZE];
//__constant__ int d_s_sub_index[6*threadNum*blockNum];
__global__ void FEpath(
//		hipComplex *d_R,  //upper triangular matrix after cholesky factorization
//		hipComplex *d_constant_shat,  //unconstrained estimation of transmitted symbol vector s
//		hipComplex *s_matrix_share,
		hipComplex *s_potential,   //the matrix use to store all the solution candidates from all the blocks
        hipComplex *s_potential_matrix,
		int *s_sub_index,   //full factorial index matrix
		int rho,
		int pathNum,
//		hipComplex *s,  //decoding results
		float *Eu,  //Euclidean distance
//		int pitch_R,    //the number of transmit antennas
//		int pitch_index,    //the number of receive antennas
//		int pitch_p,
		int M,    //modulation scheme
//		int threadNum,    //number of threads
//		int *d_list,     //the permutation list
//		hipComplex *d_psymbolconstellation,
		int index





		)
{
	//need to consider the resource allocation
	int tx=blockIdx.x*blockDim.x+threadIdx.x;     //if the path number is small we can allocate the kernel into one block so that we can use the shared memory
int tid=threadIdx.x;
int bid=blockIdx.x;
int Nt=MATRIX_SIZE;

//allocate shared memory
//	extern __shared__ hipComplex array[];

	error_t error;
	int count1, count2,count3,count4;
	__shared__ float d;    //the minimum distance unit between the signal constellation, the distance is usually 2d
	__shared__ hipComplex alpha, beta;
	alpha.x=1;alpha.y=0; beta.x=0; beta.y=0;
//	__shared__ hipComplex d_constant_shat[MATRIX_SIZE];
//	__shared__ hipComplex d_R[MATRIX_SIZE*MATRIX_SIZE];
//    __shared__ hipComplex s_temp[threadNum];
	hipComplex s_temp;
	hipComplex Eu_norm_share;
//   __shared__ hipComplex Eu_norm_share[threadNum];
    hipComplex *R_Eu_share=(hipComplex*)malloc(Nt*sizeof(hipComplex));
//    __shared__ float Eu_t[threadNum];
//	float Eu_t;
//  __shared__ hipComplex s_potential_matrix[MATRIX_SIZE*threadNum];
//    if(tid>=0&&tid<MATRIX_SIZE)
//    {
//
//		for(count1=0;count1<MATRIX_SIZE;count1++)
//		{
//			d_R[IDC2D(count1,tid,MATRIX_SIZE)]=d_R[IDC2D(count1,tid,MATRIX_SIZE)];
//		}
//
//
//		d_constant_shat[tid]=d_constant_shat[tid];
//    }

//	__syncthreads();

//	Eu_t=0;
if(index*threadNum*blockNum+tx<pathNum)
{
for (count1=Nt-1; count1>=0; count1--)
{

		if (count1<Nt-rho)
		{
			s_temp=d_constant_shat[count1];
            #pragma unroll
			for (count2=count1+1;count2<Nt; count2++)
			{
				s_temp=complex_add(s_temp,complex_mulcom(complex_div(d_R[IDC2D(count1,count2,MATRIX_SIZE)],d_R[IDC2D(count1,count1,MATRIX_SIZE)]),(complex_sub(d_constant_shat[count2],s_potential_matrix[IDC2D(count2,index*blockNum*threadNum+tx,pathNum)]))));
			}
	if(M==2)   //BPSK
	{

    	 d=sqrt(float(float(1)/float(Nt)));
    	 if(s_temp.x>0)
    	 {
    		  s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=d;
    		  s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=0;
    	 }
    	 else
    	 {
    		  s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=(-d);
    		  s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=0;
    	 }
	}
	else if(M==4)   //QPSK
	{
		int *result=(int*)malloc(sizeof(int));

		memset(result,0,sizeof(int));
		float *distance=(float*)malloc(M*sizeof(float));
		memset(distance,0,M*sizeof(int));
	    	d=sqrt(float(float(1)/float(Nt)));
           for(count2=0;count2<M;count2++)
           {
        	   switch(count2)
        	   {
        	   case 0:
        	   distance[count2]=sqrt(powf(s_temp.x-(-d),2)+pow(s_temp.y-0,2)); break;
        	   case 1:
        	   distance[count2]=sqrt(powf(s_temp.x-0,2)+pow(s_temp.y-(-d),2)); break;
        	   case 2:
        	   distance[count2]=sqrt(pow(s_temp.x-(d),2)+pow(s_temp.y-0,2)); break;
        	   case 3:
        	   distance[count2]=sqrt(pow(s_temp.x-0,2)+pow(s_temp.y-d,2)); break;
               default:
                   #if __CUDA_ARCH__ >=300
            	   printf("result error code %d\n", error);
                   #endif
            	   break;
        	   }
           }
           float  mini_distance;
  	     int mini_index;
  	     mini_distance=distance[0];
  	     mini_index=1;
  	     for(count3=0;count3<M;count3++)
  	     {
  	    	if(distance[count3]<mini_distance)
  	    	{
  	    		mini_distance=distance[count3];
  	    		mini_index=count3+1;
  	    	}
  	     }

           switch (mini_index)
           {
           case 1:
        	 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=-d;
        	 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=0;
        	break;
           case 2:
           	 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=0;
           	 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=-d;
           	break;
           case 3:
           	 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=d;
           	 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=0;
           	break;
           case 4:
       	   s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=0;
       	   s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=d;
       	  break;
           default:
               #if __CUDA_ARCH__ >=300
        	   printf("result error code %d\n", error);
               #endif
           }
           free(distance);
           free(result);
	}
	else if(M==16)  //16QAM
	{

	    	 d=sqrt(float(3)/(2* (float)(Nt*(M-1))));
	     if(s_temp.x<(-2*d))
	    	{
	    	   s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=(-3*d);
	    	}
	    else if(s_temp.x>(2*d))
	    	{
	    	   s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=(3*d);
	    	}
	    else if(s_temp.x>=0&&s_temp.x<=2*d)
	    	{
	    	   s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=d;
	    	}
	    else if(s_temp.x>=(-2*d)&&s_temp.x<=0)
	        {
	    	   s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].x=(-d);
	    	}




	    if(s_temp.y<(-2*d))
	    	{
	    	    s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=(-3*d);
	    	}
	   else if(s_temp.y>(2*d))
	    	 {
	    	   s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=(3*d);
	         }
	   else if(s_temp.y>=0&&s_temp.y<=(2*d))
	        {
	    	 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=d;
	    	}
	    else if(s_temp.y>=(-2*d)&&s_temp.y<=0)
	       {
	         s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=(-d);
	       }
//	    	 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)].y=(-d);
	}
	else if(M==64)   //64QAM	__shared__ float Eu_mini_value;
		{

	    	 d=sqrt(3/(2* (float)(Nt*(M-1))));
	     }
		}
	else
	{
		 s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)]=d_psymbolconstellation[s_sub_index[IDC2D((Nt-count1-1),(index*blockNum*threadNum+tx),pathNum)]];

	}
		R_Eu_share[count1]=complex_sub( s_potential_matrix[IDC2D(count1,index*blockNum*threadNum+tx,pathNum)],d_constant_shat[count1]);
		Eu_norm_share=beta;
        #pragma unroll
		for(count3=count1;count3<MATRIX_SIZE;count3++)
		{
//		Eu_norm_share=complex_add(Eu_norm_share,complex_mulcom(d_R[IDC2D(count1,count3,MATRIX_SIZE)],R_Eu_share[count3]));
			Eu_norm_share=complex_add(Eu_norm_share,complex_mulcom(d_R[IDC2D(count1,count3,MATRIX_SIZE)],R_Eu_share[count3]));
		}
		Eu[tx+index*blockNum*threadNum]=Eu[tx+index*blockNum*threadNum]+pow(Eu_norm_share.x,2)+pow(Eu_norm_share.y,2);

	}
//Eu[tx+index*blockNum*threadNum]=Eu_t;
__syncthreads();
free(R_Eu_share);

if(bid==0)
{
	__shared__ float Eu_mini_value_temp;
	__shared__ int Eu_mini_index;

if(tid==0)
{
	Eu_mini_value_temp=Eu[0];
    Eu_mini_index=0;
for(count1=index*blockNum*threadNum;count1<(index+1)*blockNum*threadNum;count1++)
{
	if(Eu[count1]<Eu_mini_value_temp)
	{
		Eu_mini_index=count1;
		Eu_mini_value_temp=Eu[count1];
	}
//	Eu_mini=Eu_mini_value;16
}
//printf("the temp is %d:\n", temp);
//printf("the Eu_mini_value is %f:\n", Eu_mini_value[0]);
//printf("the Eu_mini_value_temp is %f:\n", Eu_mini_value_temp);
//printf("the Eu[0] is %f:\n", Eu[0]);
//printf("the Eu_mini_index is %d:\n", Eu_mini_index);
//for(count1=0;count1<3*threadNum;count1++)
//{
//printf("%f ", Eu[count1]);
//}
}
__syncthreads();

if(tid>=0&&tid<MATRIX_SIZE)
{
	s_potential[d_list[tid]-1]=s_potential_matrix[IDC2D((MATRIX_SIZE-tid-1),Eu_mini_index,pathNum)];
//	s_potential[d_list[tid]-1]=s_potential_matrix[IDC2D((MATRIX_SIZE-tid-1),1,pathNum)];
//	printf("Eu_mini_index is %d", index);
//	printf("the Eu_mini_value is %f16:\n", Eu_mini_value[0]);
}

__syncthreads();

}
}

}

//host

void FCSD_decoding(
		hipComplex *R,  //upper triangular matrix after cholesky factorization store in device side
//		hipComplex *s_sub, //the sub brute force rho vector matrix
		hipComplex *d_s_hat,  //unconstrained estimation of transmitted symbol vector s
		hipComplex *s_kernel,  //quantization of estimation ,decoding results
//		hipComplex *Eu,  //Euclidean distance
		int Nt,    //the number of transmit antennas
		int Nr,    //the number of receive antennas
		int M,    //modulation scheme
		int *list,   //the permutation list
		hipComplex *psymbolconstellation //the symbol constellation
		)
{
//brute force search determine the vector results of the full expansion
	int rho=ceil(sqrt(Nt)-1);
	int count1,count2;
//	hipComplex *ss;
//	ss=(hipComplex*)malloc(MATRIX_SIZE*sizeof(hipComplex));
//	hipComplex *s_sub;
//	s_sub=(hipComplex*)malloc(pow(M,rho)*rho*sizeof(hipComplex));   //all the possible full expansion sub vector
	int  pathNum=pow(M,rho);
	int *Eu_mini_index=(int*)malloc(sizeof(int));
//	int *d_s_sub_index;
//	int *s_sub_index=(int*)calloc(1,rho*pow(M,rho)*sizeof(int));
	int *s_sub_index,*d_s_sub_index;
//	s_sub_index=(int*)malloc(rho*pathNum*sizeof(int));
	hipHostAlloc((void**) &s_sub_index,rho*pathNum*sizeof(int),hipHostMallocDefault);
	fullfact(rho,M,s_sub_index);    //get  the indexes of all the possible rho length symbol vectors
//	int blockNum=BLOCK_NUM;   //determined by the path number
//	int pathNum=pow(M,rho);  //number of search path
//	int threadNum=ceil(pathNum/(blockNum*stride)); //determined by the path number
    float *Eu,*d_Eu;
    hipComplex *s_potential_matrix,*d_s_potential_matrix,*d_s_kernel;
//	Eu=(float*)calloc(1,blockNum*sizeof(float));
    hipHostAlloc((void**) &Eu,blockNum*sizeof(float),hipHostMallocDefault);
    hipHostAlloc((void**) &s_potential_matrix,MATRIX_SIZE*blockNum*sizeof(hipComplex),hipHostMallocDefault);
//	hipComplex *s_potential_matrix=(hipComplex*)calloc(1,pathNum*Nt*sizeof(hipComplex));
//	hipComplex *s_hat=(hipComplex*)calloc(1,Nt*sizeof(hipComplex));
//	hipComplex  *d_s_potential_matrix;
//	hipMemcpy(s_hat,d_s_hat,Nt*sizeof(MATRIX_SIZE),hipMemcpyDeviceToHost);

//	int *j;
//	j=(int*)malloc(sizeof(int));
	hipblasHandle_t handle;
		hipblasStatus_t ret;
		hipError_t error;
//		size_t pitch_R,pitch_potential,pitch_index;
		ret=hipblasCreate(&handle);
//	    error=hipMalloc((void**) &d_R, MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
	    error=hipMalloc((void**) &d_s_sub_index, rho*pathNum*sizeof(int));
		error=hipMalloc((void**) &d_s_potential_matrix, pathNum*Nt*sizeof(hipComplex));
		hipMemset(d_s_potential_matrix,0,pathNum*Nt*sizeof(hipComplex));
//		error=hipMalloc((void**) &d_list, MATRIX_SIZE*sizeof(int));
		error=hipMalloc((void**) &d_Eu, pathNum*sizeof(float));
		hipMemset(d_Eu,0,pathNum*sizeof(float));
        error=hipMalloc((void**) &d_s_kernel, MATRIX_SIZE*sizeof(hipComplex));
//        error=hipMalloc((void**) &d_psymbolconstellation, M*sizeof(hipComplex));
        clock_t start, end;

//        hipComplex *R_constant=(hipComplex*)calloc(1,MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
    	hipMemcpyToSymbol(HIP_SYMBOL(d_R), R, MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex),0,hipMemcpyDeviceToDevice);
//    	printf("%s\n",hipGetErrorString(hipGetLastError()));
    	hipMemcpyToSymbol(HIP_SYMBOL(d_psymbolconstellation), psymbolconstellation, M*sizeof(hipComplex),0,hipMemcpyHostToDevice);
//    	printf("%s\n",hipGetErrorString(hipGetLastError()));
    	hipMemcpyToSymbol(HIP_SYMBOL(d_constant_shat), d_s_hat, MATRIX_SIZE*sizeof(hipComplex),0,hipMemcpyDeviceToDevice);
//    	printf("%s\n",hipGetErrorString(hipGetLastError()));
    	hipMemcpyToSymbol(HIP_SYMBOL(d_list), list, MATRIX_SIZE*sizeof(int),0,hipMemcpyHostToDevice);
//    	hipMemcpyToSymbol(HIP_SYMBOL(d_s_sub_index), s_sub_index, rho*pathNum*sizeof(int),0,hipMemcpyHostToDevice);
//    	printf("%s\n",hipGetErrorString(hipGetLastError()));
//        error=hipMemcpy(d_R,R,MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex),hipMemcpyDeviceToDevice);
//		error=hipMemcpy(d_psymbolconstellation, psymbolconstellation, M*sizeof(hipComplex),hipMemcpyHostToDevice);
//		error=hipMemcpy(d_s_sub_index, s_sub_index,rho*pathNum*sizeof(int),hipMemcpyHostToDevice);
//		error=hipMemcpy(d_list, list, Nt*(sizeof(int)),hipMemcpyHostToDevice);
	 int sharedMem;
    sharedMem=1*sizeof(hipComplex);
   float duration;
   float *Eu_mini;
   hipMalloc((void**) &Eu_mini,sizeof(float));
   hipMemset(Eu_mini,0,sizeof(float));
   start=clock();
   hipMemcpyAsync(d_s_sub_index,s_sub_index,rho*(pathNum)*sizeof(int),hipMemcpyHostToDevice,0);
	printf("%s\n",hipGetErrorString(hipGetLastError()));

   for(count1=0;count1<stride;count1++)
   {


//	hipMemcpyAsync(s_potential_matrix,d_s_potential_matrix,MATRIX_SIZE*(pathNum)*sizeof(hipComplex),hipMemcpyHostToDevice,0);
	FEpath<<<blockNum, threadNum,0,0>>>(d_s_kernel,d_s_potential_matrix,d_s_sub_index,rho,pathNum, d_Eu, M,count1);
	 error=hipDeviceSynchronize();
//	 hipblasIsamin(handle,blockNum,d_Eu,1,Eu_mini_index);
//	hipMemcpyAsync(s_potential_matrix,d_s_potential_matrix,MATRIX_SIZE*(blockNum)*sizeof(hipComplex),hipMemcpyDeviceToHost,0);
	printf("%s\n",hipGetErrorString(hipGetLastError()));

//    for(count2=0;count2<Nt;count2++)
//    {
//     s_kernel[list[count2]-1]=s_potential_matrix[IDC2D((MATRIX_SIZE-count2-1),(*Eu_mini_index-1),blockNum)];
//    }
//	hipMemcpyAsync(Eu,d_Eu,(pathNum)*sizeof(float),hipMemcpyDeviceToHost,0);

   }
   hipFree(Eu_mini);
   hipMemcpy(s_kernel,d_s_kernel,MATRIX_SIZE*sizeof(hipComplex),hipMemcpyDeviceToHost);


//	error=hipMemcpy(s_potential_matrix,d_s_potential_matrix, Nt*sizeof(hipComplex)*pathNum,hipMemcpyDeviceToHost);
	end=clock();
//	hipProfilerStop();
	duration=double(end-start)/double(CLOCKS_PER_SEC);

	printf("hey %0.4f ", duration);
	printf("\n");
//    memcpy(Eu+0,Eu1,pathNum*sizeof(float));
//    memcpy(Eu+pathNum,Eu2,pathNum*sizeof(float));
//    memcpy(Eu+pathNum/2,Eu3,pathNum*sizeof(float));
//    memcpy(Eu+(pathNum*3)/4,Eu4,pathNum*sizeof(float));

//	printf("Eu_num is %d", Eu_num);
//    error=hipMemcpy(s_potential_matrix,d_s_potential_matrix,pathNum*Nt*sizeof(hipComplex),hipMemcpyDeviceToHost);

//    printf("all the potential symbol vector is:\n");
//    for(count1=0;count1<blockNum;count1++)
//    {
//    	for(int count2=0;count2<Nt;count2++)
//    	{
//    		printf("%0.4f%+0.4fi ", s_potential_matrix[IDC2D(count2,count1,blockNum)].x,s_potential_matrix[IDC2D(count2,count1,blockNum)].y);
//    	}
//    	printf("\n");
//    }

//			    printf("the s_kernel is :\n");
//			    for(count1=0;count1<Nt;count1++)
//			    {
//			    	printf("%0.4f%+0.4fi ", s_kernel[count1].x, s_kernel[count1].y);
//			    }
//			    printf("\n");

			   	hipHostFree(s_sub_index);
			   	hipFree(d_s_sub_index);
//			   	hipFree(d_list);
			   	hipHostFree(Eu);
			   	hipFree(d_Eu);
			   	hipHostFree(s_potential_matrix);
			   	hipFree(d_s_potential_matrix);
			   	hipFree(d_s_kernel);
//			   	hipFree(d_psymbolconstellation);
			   	free(Eu_mini_index);
//			   	hipFree(d_R);



}

