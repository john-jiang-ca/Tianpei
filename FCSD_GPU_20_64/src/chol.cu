#include "hip/hip_runtime.h"
/*
 /*
 *  THIS FUNCTION INPLEMENT CHOLESKY FACTORIZATION
 *
 */
//#include <assert.h>
//includes system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<math.h>
//#include<hip/hip_runtime.h>
// includes CUDA
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
//#include<cutil.h>
//includes project
//#include<hip/hip_runtime_api.h>
//#include<helper_functions.h>
#include"cu_complex_operation.cuh"
#include"common.h"
#include<hip/hip_complex.h>
#include<time.h>
#include<hip/hip_runtime_api.h>
//#define BLOCKNUM 32
 __global__ void chol_kernel_optimized
 (
		 hipComplex *R,
//		 hipComplex *pivot,   //using for the data communication among all the blocks
		 int index
 )
 {
//	 int MATRIX_SIZE=N1;
	 int count1,count2;
	 int tid=threadIdx.x;
      __shared__ hipComplex update;
      extern __shared__ hipComplex array[];
	 hipComplex *matrixColumn=array;
	 __shared__  hipComplex zero;
	 zero.x=0;
	 zero.y=0;
	 //cholesky factorization row by row
//	 for (count1=0;count1<MATRIX_SIZE;count1++)
//	 {
//		 R[tid]=matrix[IDC2D(count1,tid,MATRIX_SIZE)];
//transfer the data into share array in one block
	 if(tid>=0&&tid<(MATRIX_SIZE-index))
	 {
	 for(count1=0;count1<=index;count1++)
	 {
          matrixColumn[IDC2D(count1,tid,(MATRIX_SIZE-index))]=R[IDC2D(count1,(tid+index),MATRIX_SIZE)];


	 }
//	 printf("this is working!!");
	 }
 __syncthreads();
// printf("%d ", tid);
// if(tid==0)
// {
//	 printf("hey I am here!!");
//	 for(count1=0;count1<=index;count1++)
//	 {
//		 for(count2=0;count2<MATRIX_SIZE;count2++)
//		 {
//          printf("%0.4f%+0.4fi:",R[IDC2D(count1,count2,(MATRIX_SIZE))].x,R[IDC2D(count1,count2,(MATRIX_SIZE))].y );
//		 }
//		 printf("\n");
//	 }
//}
// __syncthreads();
 //pivoting step
			if(tid==0)
			{
				if(index==0)
				{

					matrixColumn[IDC2D(0,0,(MATRIX_SIZE-index))].x=sqrt(matrixColumn[IDC2D(0,0,(MATRIX_SIZE-index))].x);
					matrixColumn[IDC2D(0,0,(MATRIX_SIZE-index))].y=0;
					R[IDC2D(0,0,MATRIX_SIZE)]=matrixColumn[IDC2D(0,0,(MATRIX_SIZE-index))];
//					printf("the first element is %0.4f%+0.4fi : \n",matrixColumn[IDC2D(index,index,MATRIX_SIZE)].x,matrixColumn[IDC2D(index,index,MATRIX_SIZE)].y );

				}
				else
				{
				update=zero;
//the addition of the column per block
				for(count2=0;count2<index;count2++)
				{
			 update=complex_add(update,complex_mulcom(complex_conjugate(matrixColumn[IDC2D(count2,0,(MATRIX_SIZE-index))]),matrixColumn[IDC2D(count2,0,(MATRIX_SIZE-index))]));
				}
				matrixColumn[IDC2D(index,0,(MATRIX_SIZE-index))]=complex_sub(matrixColumn[IDC2D(index,0,(MATRIX_SIZE-index))],update);
				matrixColumn[IDC2D(index,0,(MATRIX_SIZE-index))].x=sqrt(matrixColumn[IDC2D(index,0,(MATRIX_SIZE-index))].x);
				matrixColumn[IDC2D(index,0,(MATRIX_SIZE-index))].y=0;
				R[IDC2D(index,index,MATRIX_SIZE)]=matrixColumn[IDC2D(index,0,(MATRIX_SIZE-index))];
//				pivot[blockIdx.x]=update;
//				 printf("the  %d diagonal elements is %0.4f%+0.4fi:\n",count1,R[IDC2D(count1,count1,MATRIX_SIZE)].x,R[IDC2D(count1,count1,MATRIX_SIZE)].y );
			}
			}
__syncthreads();
// update the off-diagonal elements
if(tid>0&&tid<(MATRIX_SIZE-index))
{
if(index==0)
{
	R[IDC2D(index,(tid+index),MATRIX_SIZE)]=complex_div(matrixColumn[IDC2D(index,tid,(MATRIX_SIZE-index))],complex_conjugate(matrixColumn[IDC2D(index,0,(MATRIX_SIZE-index))]));
}
else
{
for(count2=0;count2<index;count2++)
{
matrixColumn[IDC2D(index,tid,(MATRIX_SIZE-index))]=complex_sub(matrixColumn[IDC2D(index,tid,(MATRIX_SIZE-index))],complex_mulcom(complex_conjugate(matrixColumn[IDC2D(count2,0,(MATRIX_SIZE-index))]),matrixColumn[IDC2D(count2,tid,(MATRIX_SIZE-index))]));

}
R[IDC2D(index,(tid+index),MATRIX_SIZE)]=complex_div(matrixColumn[IDC2D(index,tid,(MATRIX_SIZE-index))],complex_conjugate(matrixColumn[IDC2D(index,0,(MATRIX_SIZE-index))]));

//printf("hey attention!:\n");
//printf("%0.4f%+0.4fi ",matrixColumn[IDC2D(index,tid,(pitch))].x,matrixColumn[IDC2D(index,tid,(MATRIX_SIZE-index))].y );
}
}
__syncthreads();


//				printf("the %d %d element is %0.4f%+0.4fi ", count1,tid, R[IDC2D(count1,tid,MATRIX_SIZE)].x,R[IDC2D(count1,tid,MATRIX_SIZE)].y);
//printf("I am working !!\n");
 }

__global__ void zeroing(
		hipComplex *H
		)
{
//	int MATRIX_SIZE=N1;
 int tid=threadIdx.x;
 int count1,count2;
 for(count1=0;count1<MATRIX_SIZE;count1++)
 {
 if(tid>=0&&tid<=(count1-1))
 {
	 H[IDC2D(count1,tid,(MATRIX_SIZE))].x=0;
	 H[IDC2D(count1,tid,(MATRIX_SIZE))].y=0;
 }
 }
// if(tid==0)
// {
// for(count1=0;count1<MATRIX_SIZE;count1++)
// {
//	 for(count2=0;count2<MATRIX_SIZE;count2++)
//	 {
//		 printf("%0.4f%+0.4fi ", H[IDC2D(count1,count2,(MATRIX_SIZE))].x, H[IDC2D(count1,count2,(MATRIX_SIZE))].y);
//	 }
//	 printf("\n");
// }
// }
}
void chol(hipComplex *d_U

)
{


//	int MATRIX_SIZE=N1;
	//int m=U->size2;
//	int count;
	int count1;
	int count2;
//int MATRIX_SIZE=N1;
//allocate computation space
	hipError_t error;
//	hipComplex *d_U;
//	error=hipMalloc((void**) &d_U, MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex));
//	if(error !=hipSuccess){
//		printf("hipMalloc d_U returned error code %d, line(%d)\n", error, __LINE__);
//	}
	//data transmission from CPU to GPU


	//data transmission from CPU to GPU


//	int sharedMem=MATRIX_SIZE*MATRIX_SIZE*sizeof(hipComplex);
	//dim3 BlockId;printf("hey attention!:\n");
//	clock_t start, end;
//	start=clock();

//		int *threadID;
//		int *d_threadID;
//		hipProfilerStart();
//		threadID=(int*)malloc(sizeof(int)*threadNum);
//		hipMalloc((void**)&d_threadID, threadNum*sizeof(int));
//		hipMemcpy(d_threadID,threadID,sizeof(int)*threadNum,hipMemcpyHostToDevice);
//		error=hipMemcpy(d_U, U, sizeof(hipComplex)*MATRIX_SIZE*MATRIX_SIZE, hipMemcpyHostToDevice );
//		if(error!=hipSuccess){
//			printf("hipMemcpy U to d_U returned error code %d, line(%d)\n", error, __LINE__);
//		}
//	   double duration;
//	   clock_t start, end;
//	   start=clock();
   for(count1=0;count1<MATRIX_SIZE;count1++)
   {
		int threadNum=(MATRIX_SIZE-count1);
		int blockNum=1;
		int sharedMem=4000*sizeof(hipComplex);
		chol_kernel_optimized<<<blockNum,threadNum,sharedMem>>>(d_U,count1);
//		error=hipDeviceSynchronize();
				if(error!=hipSuccess)
				{
				printf("%s\n", hipGetErrorString(hipGetLastError()));
				}
   }
//end=clock();
//duration=double(end-start);
//printf("%0.4f ", duration);
//printf("\n");
//  start=clock();
   zeroing<<<1,MATRIX_SIZE>>>(d_U);
	error=hipDeviceSynchronize();
//	   end=clock();
//	   duration=double(end-start);
//	   printf("%0.4f ", duration);
//	   printf("\n");
			if(error!=hipSuccess)
			{
			printf("%s\n", hipGetErrorString(hipGetLastError()));
			}

//		hipError_t error;

//		hipMemcpy(threadID, d_threadID, sizeof(int)*threadNum, hipMemcpyDeviceToHost );
//		for(count1=0;count1<threadNum; count1++)
//		{
//		printf("%d", threadID[count1]);
//		printf("\n");
//		}
//		printf("%d",sizeof(hipComplex));
//		error=hipMemcpy(pR,d_U,sizeof(hipComplex)*MATRIX_SIZE*MATRIX_SIZE, hipMemcpyDeviceToHost );
//		if(error!=hipSuccess)
//		{
//			printf("hipMemcpy d_U to U returned error code %d, line(%d)\n", error, __LINE__);
//		}
//		hipProfilerStop();
//	}
//	end=clock();
//	 *durationD=(double)(end-start)/CLOCKS_PER_SEC;
	//data transmission from GPU to CPU
//	for(count1=0; count1<MATRIX_SIZE; count1++)
//	{
//		for(count2=0; count2<MATRIX_SIZE; count2++)
//		{
//			pR[count1*MATRIX_SIZE+count2]=U[count1*MATRIX_SIZE+count2];
//
//
//		}
//	}
//	U[0].x=*m;
//	pR[0].y=*m;
//    free(y);
//    free(m);
//    free(threadID);
//    hipFree(d_threadID);

}
